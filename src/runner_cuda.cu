#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of mdcore.
 * Coypright (c) 2012 Pedro Gonnet (gonnet@maths.ox.ac.uk)
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * 
 ******************************************************************************/

/* Include configuration header */
#include "../config.h"

/* Include some standard header files */
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <limits.h>

/* Include some conditional headers. */
#ifdef HAVE_MPI
    #include <mpi.h>
#endif

/* Force single precision. */
#define FPTYPE_SINGLE 1

/* Include local headers */
#include "cycle.h"
#include "errs.h"
#include "fptype.h"
#include "part.h"
#include "cell.h"
#include "space.h"
#include "potential.h"
#include "engine.h"
#include "runner.h"


/* Set the max number of parts for shared buffers. */
#define cuda_maxparts 160
#define cuda_frame 32
#define cuda_maxpots 100


/* the error macro. */
#define cuda_error(id)			( engine_err = errs_register( id , hipGetErrorString(hipGetLastError()) , __LINE__ , __FUNCTION__ , __FILE__ ) )


/* Use textured or global potential data? */
#define USETEX 1
// #define USETEX_E 1


/** Reduced part struct for CUDA. */
struct part_cuda {

    /** Particle position */
    float x[3];
    
    /** Particle force */
    float f[3];
    
    /** particle type. */
    int type;
    
    /** particle charge. */
    #ifdef USETEX_E
    float q;
    #endif
    
    };
    
    
/* The constant null potential. */
__constant__ struct potential *potential_null_cuda = NULL;

/* The number of cells and pairs. */
__constant__ int cuda_nr_pairs = 0;
__constant__ int cuda_nr_cells = 0;

/* The mutex for accessing the cell pair list. */
__device__ int cuda_cell_mutex = 0;

/* The list of cell pairs. */
__device__ struct cellpair_cuda *cuda_pairs;
__device__ int *cuda_taboo;

/* The index of the next free cell pair. */
__device__ int cuda_pair_next = 0;

/* Some constants. */
__constant__ float cuda_cutoff2 = 0.0f;
__constant__ struct potential **cuda_p;
__constant__ int *cuda_pind;
__constant__ int cuda_maxtype = 0;
__constant__ struct potential *cuda_pots;
__constant__ int *cuda_diags;

/* The potential coefficients, as a texture. */
texture< float , hipTextureType2D > tex_coeffs;
texture< float , hipTextureType2D > tex_alphas;
texture< int , hipTextureType1D > tex_offsets;
hipArray *cuda_coeffs, *cuda_alphas, *cuda_offsets;

/* Use a set of variables to communicate with the outside world. */
__device__ float cuda_fio[10];
__device__ int cuda_io[10];


/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ inline void cuda_mutex_lock ( int *m ) {
    while ( atomicCAS( m , 0 , 1 ) != 0 );
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ inline void cuda_mutex_unlock ( int *m ) {
    atomicExch( m , 0 );
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated) using
 *      texture memory on the device and explicit electrostatics.
 *
 * @param pid The index of the #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda_tex_e ( int pid , float q , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, r, ir, qir, c[potential_chunk];
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    qir = q*ir;
    
    /* compute the interval index */
    // ind = fmaxf( 0.0f , tex2D( tex_alphas , 0 , pid ) + r * ( tex2D( tex_alphas , 1 , pid ) + r * tex2D( tex_alphas , 2 , pid ) ) );
    if ( ( ind = tex2D( tex_alphas , 0 , pid ) + r * ( tex2D( tex_alphas , 1 , pid ) + r * tex2D( tex_alphas , 2 , pid ) ) ) < 0 )
        ind = 0;
    ind += tex1D( tex_offsets , pid );
    
    /* pre-load the coefficients. */
    for ( k = 0 ; k < potential_chunk ; k++ )
        c[k] = tex2D( tex_coeffs , k , ind );
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    eff = c[2];
    ee = c[2] * x + c[3];
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee + qir;
    *f = ( eff * c[1] + qir ) * ir;
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated) using
 *      texture memory on the device.
 *
 * @param pid The index of the #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda_tex ( int pid , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, r, ir, c[potential_chunk];
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    if ( ( ind = tex2D( tex_alphas , 0 , pid ) + r * ( tex2D( tex_alphas , 1 , pid ) + r * tex2D( tex_alphas , 2 , pid ) ) ) < 0 )
        ind = 0;
    ind += tex1D( tex_offsets , pid );
    
    /* pre-load the coefficients. */
    for ( k = 0 ; k < potential_chunk ; k++ )
        c[k] = tex2D( tex_coeffs , k , ind );
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    eff = c[2];
    ee = c[2] * x + c[3];
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda ( struct potential *p , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    ind = fmaxf( 0.0f , p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]) );
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param iparts_i Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param iparts_j Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 *
 * @sa #runner_dopair.
 */
 
__device__ void runner_dopair_cuda ( struct part *iparts_i , int count_i , struct part *iparts_j , int count_j, float *pshift ) {

    int k, pid, pjd, ind, wrap, threadID;
    int pjoff;
    struct part_cuda *pi, *pj;
    struct part *temp;
    #ifdef USETEX_E
        float qj, q;
    #endif
    #if defined(USETEX) || defined(USETEX_E)
        int pot;
    #else
        struct potential *pot;
    #endif
    float epot = 0.0f, dx[3], pjx[3], pjf[3], shift[3], r2, w, ee = 0.0f, eff = 0.0f;
    __shared__ struct part_cuda parts_i[ cuda_maxparts ], parts_j[ cuda_maxparts ];
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i < ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        temp = iparts_i; iparts_i = iparts_j; iparts_j = temp;
        k = count_i; count_i = count_j; count_j = k;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
    
    /* Copy the particle data to the local buffers */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        parts_i[k].x[0] = iparts_i[k].x[0];
        parts_i[k].x[1] = iparts_i[k].x[1];
        parts_i[k].x[2] = iparts_i[k].x[2];
        parts_i[k].f[0] = iparts_i[k].f[0];
        parts_i[k].f[1] = iparts_i[k].f[1];
        parts_i[k].f[2] = iparts_i[k].f[2];
        parts_i[k].type = iparts_i[k].type;
        #ifdef USETEX_E
        parts_i[k].q = iparts_i[k].q;
        #endif
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        parts_j[k].x[0] = iparts_j[k].x[0];
        parts_j[k].x[1] = iparts_j[k].x[1];
        parts_j[k].x[2] = iparts_j[k].x[2];
        parts_j[k].f[0] = iparts_j[k].f[0];
        parts_j[k].f[1] = iparts_j[k].f[1];
        parts_j[k].f[2] = iparts_j[k].f[2];
        parts_j[k].type = iparts_j[k].type;
        #ifdef USETEX_E
        parts_j[k].q = iparts_j[k].q;
        #endif
        }
        
    /* Get the wrap. */
    if ( ( wrap = count_i ) < cuda_frame )
        wrap = cuda_frame;
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = &parts_j[pjd];
        pjoff = pj->type * cuda_maxtype;
        for ( k = 0 ; k < 3 ; k++ ) {
            pjx[k] = pj->x[k] + shift[k];
            pjf[k] = 0.0f;
            }
        #ifdef USETEX_E
        qj = pj->q;
        #endif
        
        /* Loop over the particles in cell_i. */
        for ( pid = 0 ; pid < count_i ; pid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( ind = pid + threadID ) >= wrap )
                ind -= wrap;
            
            /* Do we have a pair? */
            if ( ind < count_i ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                pi = &parts_i[ ind ];
                // printf( "runner_dopair_cuda: doing pair [%i,%i].\n" , pjd , ind );

                /* Compute the radius between pi and pj. */
                for ( r2 = 0.0f , k = 0 ; k < 3 ; k++ ) {
                    dx[k] = pi->x[k] - pjx[k];
                    r2 += dx[k] * dx[k];
                    }

                /* Set the null potential if anything is bad. */
                #ifdef USETEX_E
                if ( r2 < cuda_cutoff2 && ( ( pot = cuda_pind[ pjoff + pi->type ] ) != 0 || ( q = qj*pi->q ) != 0.0f ) ) {
                #elif defined(USETEX)
                if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pjoff + pi->type ] ) != 0 ) {
                #else
                if ( r2 < cuda_cutoff2 && ( pot = cuda_p[ pjoff + pi->type ] ) != NULL ) {
                #endif

                    /* Interact particles pi and pj. */
                    #ifdef USETEX_E
                    potential_eval_cuda_tex_e( pot , q , r2 , &ee , &eff );
                    #elif defined(USETEX)
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );
                    #else
                    potential_eval_cuda( pot , r2 , &ee , &eff );
                    #endif

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pi->f[k] -= w;
                        pjf[k] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            pj->f[k] += pjf[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Copy the particle data back from the local buffers */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        iparts_i[k].f[0] = parts_i[k].f[0];
        iparts_i[k].f[1] = parts_i[k].f[1];
        iparts_i[k].f[2] = parts_i[k].f[2];
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        iparts_j[k].f[0] = parts_j[k].f[0];
        iparts_j[k].f[1] = parts_j[k].f[1];
        iparts_j[k].f[2] = parts_j[k].f[2];
        }
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param iparts_i Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param iparts_j Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 *
 * @sa #runner_dopair.
 */
 
__device__ void runner_dopair_sorted_cuda ( struct part *iparts_i , int count_i , struct part *iparts_j , int count_j, float *pshift ) {

    int k, j, i, ind, jnd, pid, pjd, pjdid, threadID, wrap;
    int pioff, swap_i;
    struct part_cuda *pi, *pj;
    struct part *temp;
    #ifdef USETEX_E
        float qi, q;
    #endif
    #if defined(USETEX) || defined(USETEX_E)
        int pot;
    #else
        struct potential *pot;
    #endif
    float epot = 0.0f, r2, w, ee = 0.0f, eff = 0.0f, inshift, swap_f, cutoff;
    float dx[3], pix[3], pif[3], shift[3];
    __shared__ struct part_cuda parts_i[ cuda_maxparts ], parts_j[ cuda_maxparts ];
    __shared__ struct {
        float d;
        int ind;
        } sort_i[ cuda_maxparts ], sort_j[ cuda_maxparts ];
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        temp = iparts_i; iparts_i = iparts_j; iparts_j = temp;
        k = count_i; count_i = count_j; count_j = k;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
        
    /* Copy the particle data to the local buffers */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        parts_i[k].x[0] = iparts_i[k].x[0];
        parts_i[k].x[1] = iparts_i[k].x[1];
        parts_i[k].x[2] = iparts_i[k].x[2];
        parts_i[k].f[0] = iparts_i[k].f[0];
        parts_i[k].f[1] = iparts_i[k].f[1];
        parts_i[k].f[2] = iparts_i[k].f[2];
        parts_i[k].type = iparts_i[k].type;
        #ifdef USETEX_E
        parts_i[k].q = iparts_i[k].q;
        #endif
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        parts_j[k].x[0] = iparts_j[k].x[0];
        parts_j[k].x[1] = iparts_j[k].x[1];
        parts_j[k].x[2] = iparts_j[k].x[2];
        parts_j[k].f[0] = iparts_j[k].f[0];
        parts_j[k].f[1] = iparts_j[k].f[1];
        parts_j[k].f[2] = iparts_j[k].f[2];
        parts_j[k].type = iparts_j[k].type;
        #ifdef USETEX_E
        parts_j[k].q = iparts_j[k].q;
        #endif
        }
        
        
    /* Pre-compute the inverse norm of the shift. */
    inshift = rsqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    cutoff = sqrt( cuda_cutoff2 );
        
    /* Pack the parts of i and j into the sort arrays. */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        sort_i[k].d = inshift * (parts_i[k].x[0]*shift[0] + parts_i[k].x[1]*shift[1] + parts_i[k].x[2]*shift[2]);
        sort_i[k].ind = k;
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        sort_j[k].d = inshift * ((shift[0]+parts_j[k].x[0])*shift[0] + (shift[1]+parts_j[k].x[1])*shift[1] + (shift[2]+parts_j[k].x[2])*shift[2]);
        sort_j[k].ind = k;
        }
    /* for ( k = count_i + threadID ; k < cuda_maxparts ; k += cuda_frame )
        sort_i[k].d = -FLT_MAX;
    for ( k = count_j + threadID ; k < cuda_maxparts ; k += cuda_frame )
        sort_j[k].d = FLT_MAX; */
        
    /* Make sure all the memory is in the right place. */
    __threadfence_block();
            
    
    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count_i ; k *= 2 ) {
        for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_i ; i += cuda_frame ) {
            jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
            if ( jnd < count_i && sort_i[ind].d < sort_i[jnd].d ) {
                swap_f = sort_i[ind].d; sort_i[ind].d = sort_i[jnd].d; sort_i[jnd].d = swap_f;
                swap_i = sort_i[ind].ind; sort_i[ind].ind = sort_i[jnd].ind; sort_i[jnd].ind = swap_i;
                }
            }
        __threadfence_block();
        for ( j = k/2 ; j > 0 ; j = j / 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_i ; i += cuda_frame ) {
                jnd = ind + j;
                if ( sort_i[ind].d < sort_i[jnd].d ) {
                    swap_f = sort_i[ind].d; sort_i[ind].d = sort_i[jnd].d; sort_i[jnd].d = swap_f;
                    swap_i = sort_i[ind].ind; sort_i[ind].ind = sort_i[jnd].ind; sort_i[jnd].ind = swap_i;
                    }
                }
            __threadfence_block();
            }
        }
    for ( k = 1 ; k < count_j ; k *= 2 ) {
        for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_j ; i += cuda_frame ) {
            jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
            if ( jnd < count_j && sort_j[ind].d > sort_j[jnd].d ) {
                swap_f = sort_j[ind].d; sort_j[ind].d = sort_j[jnd].d; sort_j[jnd].d = swap_f;
                swap_i = sort_j[ind].ind; sort_j[ind].ind = sort_j[jnd].ind; sort_j[jnd].ind = swap_i;
                }
            }
        __threadfence_block();
        for ( j = k/2 ; j > 0 ; j = j / 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_j ; i += cuda_frame ) {
                jnd = ind + j;
                if ( sort_j[ind].d > sort_j[jnd].d ) {
                    swap_f = sort_j[ind].d; sort_j[ind].d = sort_j[jnd].d; sort_j[jnd].d = swap_f;
                    swap_i = sort_j[ind].ind; sort_j[ind].ind = sort_j[jnd].ind; sort_j[jnd].ind = swap_i;
                    }
                }
            __threadfence_block();
            }
        }


    /* Verify that the sorting actually worked. */
    /* for ( k = threadID ; k < count_i-1 ; k += cuda_frame )
        if ( sort_i[k].d < sort_i[k+1].d )
            printf( "runner_dopair_sorted_cuda: sorting borked on entry %i of sort_i!\n" , k );
    for ( k = threadID ; k < count_j-1 ; k += cuda_frame )
        if ( sort_j[k].d > sort_j[k+1].d )
            printf( "runner_dopair_sorted_cuda: sorting borked on entry %i of sort_j!\n" , k ); */
    
    /* Dump the particles. */
    /* for ( k = threadID ; k < count_i ; k += cuda_frame )
        printf( "runner_dopair_sorted_cuda: part_i[%i] has d=%i.\n" ,
            sort_i[k].ind , (int)(sort_i[k].d*1000) );
    for ( k = threadID ; k < count_j ; k += cuda_frame )
        printf( "runner_dopair_sorted_cuda: part_j[%i] has d=%i.\n" ,
        sort_j[k].ind , (int)(sort_j[k].d*1000) ); */
        
        
    /* Set the initial wrap. */
    wrap = count_j;
    
    /* Loop over the particles in cell_j, frame-wise. */
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( wrap > 0 && sort_j[wrap-1].d - sort_i[pid & ~(cuda_frame - 1)].d > cutoff )
            wrap -= 1;
        if ( wrap == 0 )
            break;
        else if ( wrap < cuda_frame )
            wrap = cuda_frame;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        pi = &parts_i[ sort_i[pid].ind ];
        pioff = pi->type * cuda_maxtype;
        for ( k = 0 ; k < 3 ; k++ ) {
            pix[k] = pi->x[k] - shift[k];
            pif[k] = 0.0f;
            }
        #ifdef USETEX_E
        qi = pi->q;
        #endif
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd = pjdid + threadID ) >= wrap )
                pjd -= wrap;
            
            /* Do we have a pair? */
            if ( pjd < count_j ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                pj = &parts_j[ sort_j[pjd].ind ];

                /* Compute the radius between pi and pj. */
                for ( r2 = 0.0f , k = 0 ; k < 3 ; k++ ) {
                    dx[k] = pix[k] - pj->x[k];
                    r2 += dx[k] * dx[k];
                    }
                    
                /* Set the null potential if anything is bad. */
                #ifdef USETEX_E
                if ( r2 < cuda_cutoff2 && ( ( pot = cuda_pind[ pioff + pj->type ] ) != 0 || ( q = qj*pi->q ) != 0.0f ) ) {
                #elif defined(USETEX)
                if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pioff + pj->type ] ) != 0 ) {
                #else
                if ( r2 < cuda_cutoff2 && ( pot = cuda_p[ pioff + pj->type ] ) != NULL ) {
                #endif

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    /* Interact particles pi and pj. */
                    #ifdef USETEX_E
                    potential_eval_cuda_tex_e( pot , q , r2 , &ee , &eff );
                    #elif defined(USETEX)
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );
                    #else
                    potential_eval_cuda( pot , r2 , &ee , &eff );
                    #endif

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pif[k] -= w;
                        pj->f[k] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            pi->f[k] += pif[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Copy the particle data back from the local buffers */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        iparts_i[k].f[0] = parts_i[k].f[0];
        iparts_i[k].f[1] = parts_i[k].f[1];
        iparts_i[k].f[2] = parts_i[k].f[2];
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        iparts_j[k].f[0] = parts_j[k].f[0];
        iparts_j[k].f[1] = parts_j[k].f[1];
        iparts_j[k].f[2] = parts_j[k].f[2];
        }
        
    }


/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 *
 * @sa #runner_dopair.
 */
 
__device__ void runner_doself_cuda ( struct part *iparts , int count ) {

    int k, ind, wrap, pid, pjd, threadID;
    int pjoff;
    struct part_cuda *pi, *pj;
    #ifdef USETEX_E
        float qj, q;
    #endif
    #if defined(USETEX) || defined(USETEX_E)
        int pot;
    #else
        struct potential *pot;
    #endif
    float epot = 0.0f, dx[3], pjx[3], pjf[3], r2, w, ee, eff;
    __shared__ struct part_cuda parts[ cuda_maxparts ];
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Copy the particle data to the local buffers */
    for ( k = threadID ; k < count ; k += cuda_frame ) {
        parts[k].x[0] = iparts[k].x[0];
        parts[k].x[1] = iparts[k].x[1];
        parts[k].x[2] = iparts[k].x[2];
        parts[k].f[0] = iparts[k].f[0];
        parts[k].f[1] = iparts[k].f[1];
        parts[k].f[2] = iparts[k].f[2];
        parts[k].type = iparts[k].type;
        #ifdef USETEX_E
        parts[k].q = iparts[k].q;
        #endif
        }
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Loop over the particles in the cell, frame-wise. */
    for ( pjd = threadID ; pjd < count ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = &parts[pjd];
        pjoff = pj->type * cuda_maxtype;
        for ( k = 0 ; k < 3 ; k++ ) {
            pjx[k] = pj->x[k];
            pjf[k] = 0.0f;
            }
        #ifdef USETEX_E
        qj = pj->q;
        #endif
            
        /* Set the wrapping. */
        wrap = (pjd + (cuda_frame - 1)) & ~(cuda_frame - 1);
        
        /* Loop over the particles in cell_i. */
        for ( pid = 0 ; pid < wrap ; pid++ ) {
        
            /* Get the correct wrapped id. */
            if ( ( ind = pid + threadID ) >= wrap )
                ind -= wrap;
                
            /* Valid particle pair? */
            if ( ind < pjd ) {
                
                // if ( threadID == 0 )
                // printf( "runner_doself_cuda: doing pair [%i,%i].\n" , pjd , ind );

                /* Get a handle on the wrapped particle pid in cell_i. */
                pi = &parts[ ind ];

                /* Compute the radius between pi and pj. */
                for ( r2 = 0.0f , k = 0 ; k < 3 ; k++ ) {
                    dx[k] = pi->x[k] - pjx[k];
                    r2 += dx[k] * dx[k];
                    }

                /* Set the null potential if anything is bad. */
                #ifdef USETEX_E
                if ( r2 < cuda_cutoff2 && ( ( pot = cuda_pind[ pjoff + pi->type ] ) != 0 || ( q = qj*pi->q ) != 0.0f ) ) {
                #elif defined(USETEX)
                if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pjoff + pi->type ] ) != 0 ) {
                #else
                if ( r2 < cuda_cutoff2 && ( pot = cuda_p[ pjoff + pi->type ] ) != NULL ) {
                #endif

                    /* Interact particles pi and pj. */
                    #ifdef USETEX_E
                    potential_eval_cuda_tex_e( pot , q , r2 , &ee , &eff );
                    #elif defined(USETEX)
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );
                    #else
                    potential_eval_cuda( pot , r2 , &ee , &eff );
                    #endif

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pi->f[k] -= w;
                        pjf[k] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();

                    } /* range and potential? */

                } /* valid particle pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            pj->f[k] += pjf[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();

        } /* loop over the particles in cell_j. */
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Copy the particle data back from the local buffers */
    for ( k = threadID ; k < count ; k += cuda_frame ) {
        iparts[k].f[0] = parts[k].f[0];
        iparts[k].f[1] = parts[k].f[1];
        iparts[k].f[2] = parts[k].f[2];
        }
        
    }
    
    
/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 *
 * @sa #runner_dopair.
 */
 
__device__ void runner_doself_diag_cuda ( struct part *iparts , int count ) {

    int diag, k, diag_max, step, pid, pjd, threadID;
    struct part_cuda *pi, *pj;
    #ifdef USETEX_E
        float qj, q;
    #endif
    #if defined(USETEX) || defined(USETEX_E)
        int pot;
    #else
        struct potential *pot;
    #endif
    float epot = 0.0f, dx[3], r2, w[3], ee, eff;
    __shared__ struct part_cuda parts[ cuda_maxparts ];
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Copy the particle data to the local buffers */
    for ( k = threadID ; k < count ; k += cuda_frame ) {
        parts[k].x[0] = iparts[k].x[0];
        parts[k].x[1] = iparts[k].x[1];
        parts[k].x[2] = iparts[k].x[2];
        parts[k].f[0] = iparts[k].f[0];
        parts[k].f[1] = iparts[k].f[1];
        parts[k].f[2] = iparts[k].f[2];
        parts[k].type = iparts[k].type;
        #ifdef USETEX_E
        parts[k].q = iparts[k].q;
        #endif
        }
    
    /* Make sure everybody is in the same place. */
    __syncthreads();
    
    /* Step along the number of diagonal entries. */
    diag_max = count * (count - 1) / 2; step = 1;
    for ( diag = 0 ; diag < diag_max ; diag += step ) {
    
        /* is it time for this thread to step in? */
        if ( diag == threadID ) {
            step = diag;
            diag = (diag + 2) * (diag + 1) / 2 - 1;
            }
            
        /* If running, continue with the interactions. */
        if ( diag >= threadID && diag < diag_max ) {
        
            /* Increase the step if necessary. */
            if ( step < cuda_frame )
                step += 1;
    
            /* Get the location of the kth entry on the diagonal. */
            k = cuda_diags[ diag ]; // ( sqrtf( 8*diag + 1 ) - 1 ) / 2;
            pid = diag - k*(k+1)/2;
            pjd = count - 1 - k + pid;
            
            /* Get a handle on the particles. */
            pi = &parts[ pid ];
            pj = &parts[ pjd ];

            /* Compute the radius between pi and pj. */
            for ( r2 = 0.0f , k = 0 ; k < 3 ; k++ ) {
                dx[k] = pi->x[k] - pj->x[k];
                r2 += dx[k] * dx[k];
                }

            /* Set the null potential if anything is bad. */
            #ifdef USETEX_E
            if ( r2 < cuda_cutoff2 && ( ( pot = cuda_pind[ pj->type*cuda_maxtype + pi->type ] ) != 0 || ( q = qj*pi->q ) != 0.0f ) ) {
            #elif defined(USETEX)
            if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pj->type*cuda_maxtype + pi->type ] ) != 0 ) {
            #else
            if ( r2 < cuda_cutoff2 && ( pot = cuda_p[ pj->type*cuda_maxtype + pi->type ] ) != NULL ) {
            #endif

                /* Interact particles pi and pj. */
                #ifdef USETEX_E
                potential_eval_cuda_tex_e( pot , q , r2 , &ee , &eff );
                #elif defined(USETEX)
                potential_eval_cuda_tex( pot , r2 , &ee , &eff );
                #else
                potential_eval_cuda( pot , r2 , &ee , &eff );
                #endif

                /* Store the interaction force on pi and energy. */
                for ( k = 0 ; k < 3 ; k++ ) {
                    w[k] = eff * dx[k];
                    pi->f[k] -= w[k];
                    }

                /* Sync the shared memory values. */
                __threadfence_block();

                /* Store the interaction force on pj. */
                epot += ee;
                for ( k = 0 ; k < 3 ; k++ )
                    pj->f[k] += w[k];

                /* Sync the shared memory values. */
                __threadfence_block();

                } /* range and potential? */

            /* printf( "runner_doself_diag_cuda[%i]: diag=%i, step=%i, i=%i, j=%i.\n" ,
                threadID , diag , step , pid , pjd ); */

            } /* is it this thread's turn? */
    
        } /* Loop over diagonal indices. */
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Copy the particle data back from the local buffers */
    for ( k = threadID ; k < count ; k += cuda_frame ) {
        iparts[k].f[0] = parts[k].f[0];
        iparts[k].f[1] = parts[k].f[1];
        iparts[k].f[2] = parts[k].f[2];
        }
        
    }
    
    
/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus no
 * be externalized.
 */
 
int runner_bind ( hipArray *cuArray_coeffs , hipArray *cuArray_offsets , hipArray *cuArray_alphas ) {

    /* Bind the coeffs. */
    cuda_coeffs = cuArray_coeffs;
    if ( hipBindTextureToArray( tex_coeffs , cuArray_coeffs ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Bind the offsets. */
    cuda_offsets = cuArray_offsets;
    if ( hipBindTextureToArray( tex_offsets , cuArray_offsets ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Bind the alphas. */
    cuda_alphas = cuArray_alphas;
    if ( hipBindTextureToArray( tex_alphas , cuArray_alphas ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Rock and roll. */
    return runner_err_ok;

    }


/**
 * @brief Loop over the cell pairs and process them.
 *
 * @param cells Array of cells on the device.
 *
 */
 
__global__ void runner_run_cuda ( struct part *parts[] , int *counts ) {

    int blockID, threadID;
    int i;
    struct cellpair_cuda temp;
    __shared__ int finger, cid, cjd;
    
    /* Get the block and thread ids. */
    blockID = blockIdx.x;
    threadID = threadIdx.x;
    
    /* Check that we've got the correct warp size! */
    /* if ( warpSize != cuda_frame ) {
        if ( blockID == 0 && threadID == 0 )
            printf( "runner_run_cuda: error: the warp size of the device (%i) does not match the warp size mdcore was compiled for (%i).\n" ,
                warpSize , cuda_frame );
        return;
        } */
    
    /* Main loop... */
    while ( cuda_pair_next < cuda_nr_pairs ) {
    
        /* Try to catch a pair. */
        if ( threadID == 0 ) {
        
            /* Lock the mutex. */
            cuda_mutex_lock( &cuda_cell_mutex );
            
            /* Loop over the remaining pairs... */
            for ( i = cuda_pair_next ; i < cuda_nr_pairs ; i++ )
                if ( cuda_taboo[ cuda_pairs[i].i ] == 0 &&
                     cuda_taboo[ cuda_pairs[i].j ] == 0 )
                    break;
                    
            /* If we actually got a pair, flip it to the top and decrease
               cuda_pair_next. */
            if ( i < cuda_nr_pairs ) {
                temp = cuda_pairs[i];
                cuda_pairs[i] = cuda_pairs[ cuda_pair_next ];
                cuda_pairs[ cuda_pair_next ] = temp;
                finger = cuda_pair_next;
                cid = cuda_pairs[finger].i; cjd = cuda_pairs[finger].j;
                cuda_pair_next += 1;
                cuda_taboo[ cid ] = 1;
                cuda_taboo[ cjd ] = 1;
                }
            else
                finger = -1;
            
            /* Make sure everybody is on the same page. */
            __threadfence();
        
            /* Un-lock the mutex. */
            cuda_mutex_unlock( &cuda_cell_mutex );
            
            }
            
        /* Get everybody together. */
        __syncthreads();
            
        /* If we actually got a pair, do it! */
        if ( finger >= 0 ) {
        
            // if ( threadID == 0 )
            //     printf( "runner_run_cuda: block %i working on pair [%i,%i] (finger = %i).\n" , blockID , cid , cjd , finger );
        
            /* Do the pair. */
            if ( cid != cjd )
                runner_dopair_sorted_cuda( parts[cid] , counts[cid] , parts[cjd] , counts[cjd] , cuda_pairs[finger].shift );
            else
                runner_doself_diag_cuda( parts[cid] , counts[cid] );
        
            /* Release the cells in the taboo list. */
            if ( threadID == 0 ) {
                cuda_taboo[ cid ] = 0;
                cuda_taboo[ cjd ] = 0;
                __threadfence();
                }
            
            }
    
        /* Get everybody together. */
        __syncthreads();
            
        } /* main loop. */

    }

