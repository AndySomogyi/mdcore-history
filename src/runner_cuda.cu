#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of mdcore.
 * Coypright (c) 2012 Pedro Gonnet (pedro.gonnet@durham.ac.uk)
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * 
 ******************************************************************************/

/* Include configuration header */
#include "../config.h"

/* Include some standard header files */
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <limits.h>

/* Include headers for overloaded vector functions. */
#include "cutil_math.h"

/* Include some conditional headers. */
#ifdef HAVE_MPI
    #include <mpi.h>
#endif

/* Force single precision. */
#define FPTYPE_SINGLE 1

/* Disable vectorization for the nvcc compiler's sake. */
#undef __SSE__
#undef __SSE2__
#undef __ALTIVEC__
#undef __AVX__

/* Include local headers */
#include "cycle.h"
#include "errs.h"
#include "fptype.h"
#include "lock.h"
#include "part.h"
#include "cell.h"
#include "fifo.h"
#include "space.h"
#include "potential.h"
#include "engine.h"
#include "runner.h"
#include "runner_cuda.h"


/* the error macro. */
#define error(id)				( engine_err = errs_register( id , engine_err_msg[-(id)] , __LINE__ , __FUNCTION__ , __FILE__ ) )
#define cuda_error(id)			( engine_err = errs_register( id , hipGetErrorString(hipGetLastError()) , __LINE__ , __FUNCTION__ , __FILE__ ) )


/* The constant null potential. */
__constant__ struct potential *potential_null_cuda = NULL;

/* The number of cells and pairs. */
__constant__ int cuda_nr_pairs = 0;
__device__ int cuda_pairs_done = 0;
__constant__ int cuda_nr_tuples = 0;
__constant__ int cuda_nr_cells = 0;

/* The parts (non-texture access). */
__constant__ float4 *cuda_parts;

/* Diagonal entries and potential index lookup table. */
__constant__ unsigned int *cuda_diags;
__constant__ unsigned int *cuda_pind;

/* The mutex for accessing the cell pair list. */
__device__ int cuda_cell_mutex = 0;
__device__ int cuda_barrier = 0;

/* The list of cell pairs. */
__constant__ struct cellpair_cuda *cuda_pairs;
__device__ int *cuda_taboo;

/* The index of the next free cell pair. */
__device__ int cuda_pair_next = 0;

/* Indices for the "new" queue. */
__device__ int cuda_pair_count = 0;
__device__ int cuda_pair_curr = 0;
__device__ int *cuda_pairIDs;

/* Some constants. */
__constant__ float cuda_cutoff2 = 0.0f;
__constant__ float cuda_cutoff = 0.0f;
__constant__ float cuda_dscale = 0.0f;
__constant__ float cuda_maxdist = 0.0f;
__constant__ struct potential **cuda_p;
__constant__ int cuda_maxtype = 0;
__constant__ struct potential *cuda_pots;

/* Sortlists for the Verlet algorithm. */
__device__ unsigned int *cuda_sortlists = NULL;
__device__ int *cuda_sortlists_ind;

/* The potential coefficients, as a texture. */
texture< float4 , hipTextureType2D > tex_coeffs;
texture< float4 , hipTextureType2D > tex_parts;

/* Other textures. */
texture< int , hipTextureType1D > tex_pind;
texture< unsigned int , hipTextureType1D > tex_diags;

/* Arrays to hold the textures. */
hipArray *cuda_coeffs;

/* The potential parameters (hard-wired size for now). */
__constant__ float cuda_eps[ 100 ];
__constant__ float cuda_rmin[ 100 ];

/* The list of fifos to work with. */
__device__ struct fifo_cuda cuda_fifos_in[ cuda_maxblocks ];
__device__ struct fifo_cuda cuda_fifos_out[ cuda_maxblocks ];

/* Use a set of variables to communicate with the outside world. */
__device__ float cuda_fio[32];
__device__ int cuda_io[32];
__device__ int cuda_rcount = 0;

/* Potential energy. */
__device__ float cuda_epot = 0.0f, cuda_epot_out;

/* Timers. */
__device__ float cuda_timers[ tid_count ];


/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ void cuda_mutex_lock ( int *m ) {
    TIMER_TIC
    while ( atomicCAS( m , 0 , 1 ) != 0 );
    TIMER_TOC( tid_mutex )
    }


/**
 * @brief Attempt to lock a device mutex.
 *
 * @param m The mutex.
 *
 * Try to grab the mutex. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_trylock ( int *m ) {
    TIMER_TIC
    return atomicCAS( m , 0 , 1 ) == 0;
    TIMER_TOC( tid_mutex )
    }


/**
 * @brief Lock a device mutex with an additional condition.
 *
 * @param m The mutex.
 * @param c the condition
 *
 * @return @c 1 if the mutex could be locked or zero if the condition @c c
 * was reached first.
 *
 * Loops until the mutex can be set or until @c *c is non-zero.
 * Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_lock_cond ( int *m , int *c ) {
    TIMER_TIC
    while ( atomicCAS( c , 0 , 0 ) == 0 )
        if ( atomicCAS( m , 0 , 1 ) == 0 ) {
            TIMER_TOC( tid_mutex )
            return 1;
            }
    TIMER_TOC( tid_mutex )
    return 0;
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ void cuda_mutex_unlock ( int *m ) {
    atomicExch( m , 0 );
    }
    
    
/**
 * @brief Push an element onto a #fifo_cuda, blocking.
 *
 * @return The number of elements in the #fifo_cuda.
 */
 
__device__ inline int cuda_fifo_push ( struct fifo_cuda *f , unsigned int e ) {

    /* Wait for there to be space in the list. */
    while ( f->count == cuda_fifo_size );

    /* Put the element in the list. */
    atomicExch( &(f->data[ f->last ]) , e );
    
    /* Increase the "last" counter. */
    atomicExch( &f->last , (f->last + 1) % cuda_fifo_size );
    
    /* Increase the count. */
    atomicAdd( &f->count , 1 );
    
    /* Return the fifo size. */
    return f->count;

    }
    
    
/**
 * @brief Pop an element from a #fifo_cuda, blocking.
 *
 * @return The popped element.
 */
 
__device__ inline unsigned int cuda_fifo_pop ( struct fifo_cuda *f ) {

    /* Wait for there to be something in the fifo. */
    while ( f->count == 0 );

    unsigned int res = f->data[ f->first ];

    /* Increase the "fist" counter. */
    atomicExch( &f->first , (f->first + 1) % cuda_fifo_size );
    
    /* Decrease the count. */
    atomicSub( &f->count , 1 );
    
    /* Return the first element. */
    return res;

    }
    
    
/**
 * @brief Copy bulk memory in a strided way.
 *
 * @param dest Pointer to destination memory.
 * @param source Pointer to source memory.
 * @param count Number of bytes to copy, must be a multiple of sizeof(int).
 */
 
__device__ inline void cuda_memcpy ( void *dest , void *source , int count ) {

    int j, k, icount = count / sizeof(int) / cuda_frame / cuda_memcpy_chunk;
    int *idest = (int *)dest, *isource = (int *)source;
    int chunk[cuda_memcpy_chunk];
    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 ; k < icount ; k += 1 ) {
        #pragma unroll
        for ( j = 0 ; j < cuda_memcpy_chunk ; j++ )
            chunk[j] = isource[ (cuda_memcpy_chunk*k+j)*cuda_frame + threadID ];
        #pragma unroll
        for ( j = 0 ; j < cuda_memcpy_chunk ; j++ )
            idest[ (cuda_memcpy_chunk*k+j)*cuda_frame + threadID ] = chunk[j];
        }
    for ( k = cuda_memcpy_chunk*cuda_frame*icount + threadID ; k < count/sizeof(int) ; k += cuda_frame )
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
    }
    
    
__device__ inline void cuda_memcpy_old ( void *dest , void *source , int count ) {

    int k;
    volatile int *idest = (int *)dest, *isource = (int *)source;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 + threadIdx.x ; k < count/sizeof(int) ; k += cuda_frame )
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
    }


/**
 * @brief Sum two vectors in a strided way.
 *
 * @param a Pointer to destination memory.
 * @param b Pointer to source memory.
 * @param count Number of floats to sum.
 *
 * Computes @c a[k] += b[k] for k=1..count.
 */
 
__device__ inline void cuda_sum ( float *a , float *b , int count ) {

    int i, j, k, icount = count / cuda_frame / cuda_sum_chunk;
    float chunk[cuda_memcpy_chunk];
    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 ; k < icount ; k += 1 ) {
        #pragma unroll
        for ( j = 0 ; j < cuda_sum_chunk ; j++ ) {
            i = (cuda_sum_chunk*k+j)*cuda_frame + threadID;
            chunk[j] = a[i] + b[i];
            }
        #pragma unroll
        for ( j = 0 ; j < cuda_sum_chunk ; j++ )
            a[ (cuda_sum_chunk*k+j)*cuda_frame + threadID ] = chunk[j];
        }
    for ( k = cuda_sum_chunk*cuda_frame*icount + threadID ; k < count ; k += cuda_frame )
        a[k] += b[k];
        
    TIMER_TOC(tid_update)
        
    }
    
    
/**
 * @brief Sort the given data w.r.t. the lowest 16 bits in decending order.
 *
 * @param a The array to sort.
 * @param count The number of elements.
 */
 
__device__ void cuda_sort_descending ( unsigned int *a , int count ) {

    int i, j, k, threadID = threadIdx.x;
    int hi[2], lo[2], ind[2], jnd[2];
    unsigned int swap_i[2], swap_j[2];

    TIMER_TIC

    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count ; k *= 2 ) {
    
        /* First step. */
        for ( i = threadID ;  i < count ; i += 2*cuda_frame ) {
            hi[0] = i & ~(k-1); lo[0] = i & (k-1);
            hi[1] = (i + cuda_frame) & ~(k-1); lo[1] = (i + cuda_frame) & (k-1);
            ind[0] = i + hi[0]; jnd[0] = 2*(hi[0]+k) - lo[0] - 1;
            ind[1] = i + cuda_frame + hi[1]; jnd[1] = 2*(hi[1]+k) - lo[1] - 1;
            swap_i[0] = ( jnd[0] < count ) ? a[ind[0]] : 0;
            swap_i[1] = ( jnd[1] < count ) ? a[ind[1]] : 0;
            swap_j[0] = ( jnd[0] < count ) ? a[jnd[0]] : 0;
            swap_j[1] = ( jnd[1] < count ) ? a[jnd[1]] : 0;
            if  ( ( swap_i[0] & 0xffff ) < ( swap_j[0] & 0xffff ) ) {
                a[ind[0]] = swap_j[0];
                a[jnd[0]] = swap_i[0];
                }
            if  ( ( swap_i[1] & 0xffff ) < ( swap_j[1] & 0xffff ) ) {
                a[ind[1]] = swap_j[1];
                a[jnd[1]] = swap_i[1];
                }
            }
            
        /* Let that last step sink in. */
        // __threadfence_block();
    
        /* Second step(s). */
        for ( j = k/2 ; j > 0 ; j /= 2 ) {
            for ( i = threadID ;  i < count ; i += 2*cuda_frame ) {
                hi[0] = i & ~(j-1);
                hi[1] = (i + cuda_frame) & ~(j-1);
                ind[0] = i + hi[0]; jnd[0] = ind[0] + j;
                ind[1] = i + cuda_frame + hi[1]; jnd[1] = ind[1] + j;
                swap_i[0] = ( jnd[0] < count ) ? a[ind[0]] : 0;
                swap_i[1] = ( jnd[1] < count ) ? a[ind[1]] : 0;
                swap_j[0] = ( jnd[0] < count ) ? a[jnd[0]] : 0;
                swap_j[1] = ( jnd[1] < count ) ? a[jnd[1]] : 0;
                if  ( ( swap_i[0] & 0xffff ) < ( swap_j[0] & 0xffff ) ) {
                    a[ind[0]] = swap_j[0];
                    a[jnd[0]] = swap_i[0];
                    }
                if  ( ( swap_i[1] & 0xffff ) < ( swap_j[1] & 0xffff ) ) {
                    a[ind[1]] = swap_j[1];
                    a[jnd[1]] = swap_i[1];
                    }
                }
            // __threadfence_block();
            }
            
        }
        
    TIMER_TOC(tid_sort)
        
    }

    
    
/**
 * @brief Sort the given data w.r.t. the lowest 16 bits in ascending order.
 *
 * @param a The array to sort.
 * @param count The number of elements.
 */
 
__device__ void cuda_sort_ascending ( unsigned int *a , int count ) {

    int i, j, k, threadID = threadIdx.x;
    int hi[2], lo[2], ind[2], jnd[2];
    unsigned int swap_i[2], swap_j[2];

    TIMER_TIC

    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count ; k *= 2 ) {
    
        /* First step. */
        for ( i = threadID ;  i < count ; i += 2*cuda_frame ) {
            hi[0] = i & ~(k-1); lo[0] = i & (k-1);
            hi[1] = (i + cuda_frame) & ~(k-1); lo[1] = (i + cuda_frame) & (k-1);
            ind[0] = i + hi[0]; jnd[0] = 2*(hi[0]+k) - lo[0] - 1;
            ind[1] = i + cuda_frame + hi[1]; jnd[1] = 2*(hi[1]+k) - lo[1] - 1;
            swap_i[0] = ( jnd[0] < count ) ? a[ind[0]] : 0;
            swap_i[1] = ( jnd[1] < count ) ? a[ind[1]] : 0;
            swap_j[0] = ( jnd[0] < count ) ? a[jnd[0]] : 0;
            swap_j[1] = ( jnd[1] < count ) ? a[jnd[1]] : 0;
            if  ( ( swap_i[0] & 0xffff ) > ( swap_j[0] & 0xffff ) ) {
                a[ind[0]] = swap_j[0];
                a[jnd[0]] = swap_i[0];
                }
            if  ( ( swap_i[1] & 0xffff ) > ( swap_j[1] & 0xffff ) ) {
                a[ind[1]] = swap_j[1];
                a[jnd[1]] = swap_i[1];
                }
            }
            
        /* Let that last step sink in. */
        // __threadfence_block();
    
        /* Second step(s). */
        for ( j = k/2 ; j > 0 ; j /= 2 ) {
            for ( i = threadID ;  i < count ; i += 2*cuda_frame ) {
                hi[0] = i & ~(j-1);
                hi[1] = (i + cuda_frame) & ~(j-1);
                ind[0] = i + hi[0]; jnd[0] = ind[0] + j;
                ind[1] = i + cuda_frame + hi[1]; jnd[1] = ind[1] + j;
                swap_i[0] = ( jnd[0] < count ) ? a[ind[0]] : 0;
                swap_i[1] = ( jnd[1] < count ) ? a[ind[1]] : 0;
                swap_j[0] = ( jnd[0] < count ) ? a[jnd[0]] : 0;
                swap_j[1] = ( jnd[1] < count ) ? a[jnd[1]] : 0;
                if  ( ( swap_i[0] & 0xffff ) > ( swap_j[0] & 0xffff ) ) {
                    a[ind[0]] = swap_j[0];
                    a[jnd[0]] = swap_i[0];
                    }
                if  ( ( swap_i[1] & 0xffff ) > ( swap_j[1] & 0xffff ) ) {
                    a[ind[1]] = swap_j[1];
                    a[jnd[1]] = swap_i[1];
                    }
                }
            // __threadfence_block();
            }
            
        }
        
    TIMER_TOC(tid_sort)
        
    }

    
    
/** 
 * @brief Evaluates the given potential at the given point (interpolated) using
 *      texture memory on the device.
 *
 * @param pid The index of the #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda_tex ( int pid , float r2 , float *e , float *f ) {

    int ind;
    float x, ee, eff, r, ir;
    float4 alpha, c1, c2;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    alpha = tex2D( tex_coeffs , 0 , pid );
    // alpha = tex1D( tex_alphas , pid );
    if ( ( ind = alpha.x + r * ( alpha.y + r * alpha.z ) ) < 0 )
        ind = 0;
    
    /* pre-load the coefficients. */
    c1 = tex2D( tex_coeffs , 2*ind+2 , pid );
    c2 = tex2D( tex_coeffs , 2*ind+3 , pid );
    
    /* adjust x to the interval */
    x = (r - c1.x) * c1.y;
    
    /* compute the potential and its derivative */
    eff = c1.z;
    ee = c1.z * x + c1.w;
    eff = eff * x + ee;
    ee = ee * x + c2.x;
    eff = eff * x + ee;
    ee = ee * x + c2.y;
    eff = eff * x + ee;
    ee = ee * x + c2.z;
    eff = eff * x + ee;
    ee = ee * x + c2.w;

    /* store the result */
    *e = ee; *f = eff * c1.y * ir;
        
    TIMER_TOC(tid_potential)
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated) using
 *      texture memory on the device.
 *
 * @param pid The index of the #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval4_cuda_tex ( int4 pid , float4 r2 , float4 *e , float4 *f ) {

    int4 ind;
    float4 x, ee, eff, r, ir, c1[4], c2[4], a[4];
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir.x = rsqrtf(r2.x);
    ir.y = rsqrtf(r2.y);
    ir.z = rsqrtf(r2.z);
    ir.w = rsqrtf(r2.w);
    r = r2*ir;
    
    /* compute the interval index */
    a[0] = tex2D( tex_coeffs , 0 , pid.x );
    a[1] = tex2D( tex_coeffs , 0 , pid.y );
    a[2] = tex2D( tex_coeffs , 0 , pid.z );
    a[3] = tex2D( tex_coeffs , 0 , pid.w );
    /* a[0] = tex1D( tex_alphas , pid.x );
    a[1] = tex1D( tex_alphas , pid.y );
    a[2] = tex1D( tex_alphas , pid.z );
    a[3] = tex1D( tex_alphas , pid.w ); */
    ind.x = max( 0 , (int)( a[0].x + r.x * ( a[0].y + r.x * a[0].z ) ) );
    ind.y = max( 0 , (int)( a[1].x + r.y * ( a[1].y + r.y * a[1].z ) ) );
    ind.z = max( 0 , (int)( a[2].x + r.z * ( a[2].y + r.z * a[2].z ) ) );
    ind.w = max( 0 , (int)( a[3].x + r.w * ( a[3].y + r.w * a[3].z ) ) );
    
    /* pre-load the coefficients. */
    c1[0] = tex2D( tex_coeffs , 2*ind.x+2 , pid.x );
    c2[0] = tex2D( tex_coeffs , 2*ind.x+3 , pid.x );
    c1[1] = tex2D( tex_coeffs , 2*ind.y+2 , pid.y );
    c2[1] = tex2D( tex_coeffs , 2*ind.y+3 , pid.y );
    c1[2] = tex2D( tex_coeffs , 2*ind.z+2 , pid.z );
    c2[2] = tex2D( tex_coeffs , 2*ind.z+3 , pid.z );
    c1[3] = tex2D( tex_coeffs , 2*ind.w+2 , pid.w );
    c2[3] = tex2D( tex_coeffs , 2*ind.w+3 , pid.w );
    
    /* adjust x to the interval */
    x.x = (r.x - c1[0].x) * c1[0].y;
    x.y = (r.y - c1[1].x) * c1[1].y;
    x.z = (r.z - c1[2].x) * c1[2].y;
    x.w = (r.w - c1[3].x) * c1[3].y;
    
    /* compute the potential and its derivative */
    eff.x = c1[0].z;
    eff.y = c1[1].z;
    eff.z = c1[2].z;
    eff.w = c1[3].z;
    ee.x = c1[0].z * x.x + c1[0].w;
    ee.y = c1[1].z * x.y + c1[1].w;
    ee.z = c1[2].z * x.z + c1[2].w;
    ee.w = c1[3].z * x.w + c1[3].w;
    eff.x = eff.x * x.x + ee.x;
    eff.y = eff.y * x.y + ee.y;
    eff.z = eff.z * x.z + ee.z;
    eff.w = eff.w * x.w + ee.w;
    ee.x = ee.x * x.x + c2[0].x;
    ee.y = ee.y * x.y + c2[1].x;
    ee.z = ee.z * x.z + c2[2].x;
    ee.w = ee.w * x.w + c2[3].x;
    eff.x = eff.x * x.x + ee.x;
    eff.y = eff.y * x.y + ee.y;
    eff.z = eff.z * x.z + ee.z;
    eff.w = eff.w * x.w + ee.w;
    ee.x = ee.x * x.x + c2[0].y;
    ee.y = ee.y * x.y + c2[1].y;
    ee.z = ee.z * x.z + c2[2].y;
    ee.w = ee.w * x.w + c2[3].y;
    eff.x = eff.x * x.x + ee.x;
    eff.y = eff.y * x.y + ee.y;
    eff.z = eff.z * x.z + ee.z;
    eff.w = eff.w * x.w + ee.w;
    ee.x = ee.x * x.x + c2[0].z;
    ee.y = ee.y * x.y + c2[1].z;
    ee.z = ee.z * x.z + c2[2].z;
    ee.w = ee.w * x.w + c2[3].z;
    eff.x = eff.x * x.x + ee.x;
    eff.y = eff.y * x.y + ee.y;
    eff.z = eff.z * x.z + ee.z;
    eff.w = eff.w * x.w + ee.w;
    ee.x = ee.x * x.x + c2[0].w;
    ee.y = ee.y * x.y + c2[1].w;
    ee.z = ee.z * x.z + c2[2].w;
    ee.w = ee.w * x.w + c2[3].w;

    /* Scale the derivative accordingly. */
    eff.x *= c1[0].y * ir.x;
    eff.y *= c1[1].y * ir.y;
    eff.z *= c1[2].y * ir.z;
    eff.w *= c1[3].y * ir.w;
    
    /* store the result */
    *e = ee; *f = eff;
        
    TIMER_TOC(tid_potential4)
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda ( struct potential *p , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    ind = fmaxf( 0.0f , p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]) );
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */

#ifdef PARTS_TEX 
__device__ void runner_dopair_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , float *pshift , float *epot_global ) {
#else
__device__ void runner_dopair_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , float *pshift , float *epot_global ) {
#endif

    int k, pid, pjd, ind, wrap_i, threadID;
    int pjoff;
    int pot;
    float epot = 0.0f, dx[3], pjf[3], shift[3], r2, w;
    float ee = 0.0f, eff = 0.0f;
    float4 pi, pj;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    /* if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i < ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        float *temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else */ {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

    /* Get the wraps. */
    wrap_i = (count_i < cuda_frame) ? cuda_frame : count_i;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cjd );
        #else
            pj = parts_j[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        pj.x += shift[0]; pj.y += shift[1]; pj.z += shift[2];
        pjf[0] = 0.0f; pjf[1] = 0.0f; pjf[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = 0 ; ind < wrap_i ; ind++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pid = ind + threadID ) >= wrap_i )
                pid -= wrap_i;
            if ( pid < count_i ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                #ifdef PARTS_TEX
                    pi = tex2D( tex_parts , pid , cid );
                #else
                    pi = parts_i[ pid ];
                #endif

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
                dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
                dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];

                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pjoff + (int)pi.w ] ) != 0 ) {

                    // atomicAdd( &cuda_rcount , 1 );
                
                    /* Interact particles pi and pj. */
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        forces_i[ 3*pid + k ] -= w;
                        pjf[k] += w;
                        }

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                    } /* in range and potential. */

                } /* valid pid? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_j[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        // __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */

#ifdef PARTS_TEX 
__device__ void runner_dopair4_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , float *pshift , float *epot_global ) {
#else
__device__ void runner_dopair4_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , float *pshift , float *epot_global ) {
#endif

    int k, pjd, ind, wrap_i, threadID;
    int pjoff;
    float4 pi[4], pj;
    int4 pot, pid;
    char4 valid;
    float4 r2, ee, eff;
    float epot = 0.0f, dx[12], pjf[3], shift[3], w;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    /* if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i < ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        float *temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else */ {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

    /* Get the wraps. */
    wrap_i = (count_i < cuda_frame) ? cuda_frame : count_i;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cjd );
        #else
            pj = parts_j[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        pj.x += shift[0]; pj.y += shift[1]; pj.z += shift[2];
        for ( k = 0 ; k < 3 ; k++ )
            pjf[k] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = 0 ; ind < wrap_i ; ind += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pid.x = ind + threadID ) >= wrap_i )
                pid.x -= wrap_i;
            if ( ( pid.y = ind + threadID + 1 ) >= wrap_i )
                pid.y -= wrap_i;
            if ( ( pid.z = ind + threadID + 2 ) >= wrap_i )
                pid.z -= wrap_i;
            if ( ( pid.w = ind + threadID + 3 ) >= wrap_i )
                pid.w -= wrap_i;
                
            /* Get the particle pointers. */
            #ifdef PARTS_TEX
                pi[0] = ( valid.x = ( pid.x < count_i ) ) ? tex2D( tex_parts , pid.x , cid ) : pj;
                pi[1] = ( valid.y = ( pid.y < count_i ) && ( ind + 1 < wrap_i ) ) ? tex2D( tex_parts , pid.y , cid ) : pj;
                pi[2] = ( valid.z = ( pid.z < count_i ) && ( ind + 2 < wrap_i ) ) ? tex2D( tex_parts , pid.z , cid ) : pj;
                pi[3] = ( valid.w = ( pid.w < count_i ) && ( ind + 3 < wrap_i ) ) ? tex2D( tex_parts , pid.w , cid ) : pj;
            #else
                pi[0] = ( valid.x = ( pid.x < count_i ) ) ? parts_i[ pid.x] : pj;
                pi[1] = ( valid.y = ( pid.y < count_i ) && ( ind + 1 < wrap_i ) ) ? parts_i[ pid.y ] : pj;
                pi[2] = ( valid.z = ( pid.z < count_i ) && ( ind + 2 < wrap_i ) ) ? parts_i[ pid.z ] : pj;
                pi[3] = ( valid.w = ( pid.w < count_i ) && ( ind + 3 < wrap_i ) ) ? parts_i[ pid.w ] : pj;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pj.x - pi[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pj.y - pi[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pj.z - pi[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pj.x - pi[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pj.y - pi[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pj.z - pi[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pj.x - pi[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pj.y - pi[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pj.z - pi[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pj.x - pi[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pj.y - pi[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pj.z - pi[3].z; r2.w += dx[11] * dx[11];

                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? cuda_pind[ pjoff + (int)pi[0].w ] : 0;
            pot.y = valid.y ? cuda_pind[ pjoff + (int)pi[1].w ] : 0;
            pot.z = valid.z ? cuda_pind[ pjoff + (int)pi[2].w ] : 0;
            pot.w = valid.w ? cuda_pind[ pjoff + (int)pi[3].w ] : 0;
            
            /* if ( pot.x != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.y != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.z != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.w != 0 )
                atomicAdd( &cuda_rcount , 1 ); */
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Update the forces. */
            if ( valid.x ) {
                pjf[0] -= ( w = eff.x * dx[0] ); forces_i[ 3*pid.x + 0 ] += w;
                pjf[1] -= ( w = eff.x * dx[1] ); forces_i[ 3*pid.x + 1 ] += w;
                pjf[2] -= ( w = eff.x * dx[2] ); forces_i[ 3*pid.x + 2 ] += w;
                epot += ee.x;
                }
            // __threadfence_block();
            if ( valid.y ) {
                pjf[0] -= ( w = eff.y * dx[3] ); forces_i[ 3*pid.y + 0 ] += w;
                pjf[1] -= ( w = eff.y * dx[4] ); forces_i[ 3*pid.y + 1 ] += w;
                pjf[2] -= ( w = eff.y * dx[5] ); forces_i[ 3*pid.y + 2 ] += w;
                epot += ee.y;
                }
            // __threadfence_block();
            if ( valid.z ) {
                pjf[0] -= ( w = eff.z * dx[6] ); forces_i[ 3*pid.z + 0 ] += w;
                pjf[1] -= ( w = eff.z * dx[7] ); forces_i[ 3*pid.z + 1 ] += w;
                pjf[2] -= ( w = eff.z * dx[8] ); forces_i[ 3*pid.z + 2 ] += w;
                epot += ee.z;
                }
            // __threadfence_block();
            if ( valid.w ) {
                pjf[0] -= ( w = eff.w * dx[9] ); forces_i[ 3*pid.w + 0 ] += w;
                pjf[1] -= ( w = eff.w * dx[10] ); forces_i[ 3*pid.w + 1 ] += w;
                pjf[2] -= ( w = eff.w * dx[11] ); forces_i[ 3*pid.w + 2 ] += w;
                epot += ee.w;
                }
            // __threadfence_block();
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_j[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        // __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair_verlet_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist , float *epot_global ) {
#else
__device__ void runner_dopair_verlet_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist , float *epot_global ) {
#endif

    int k, pid, pjd, spid, spjd, pjdid, threadID, wrap, cj;
    int pioff;
    unsigned int dmaxdist;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, r2, w, ee = 0.0f, eff = 0.0f, nshift, inshift;
    float dx[3], pif[3], shift[3], shiftn[3];
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    /* if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        float *temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else */ {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;
       
    TIMER_TIC2
        
    /* Re-build sorted pairs list? */
    if ( verlet_rebuild ) {
    
        /* Pack the parts of i and j into the sort arrays. */
        for ( k = threadID ; k < count_i ; k += cuda_frame ) {
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , k , cid );
            #else
                pi = parts_i[ k ];
            #endif
            sort_i[k] = ( k << 16 ) |
                (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
            }
        for ( k = threadID ; k < count_j ; k += cuda_frame ) {
            #ifdef PARTS_TEX
                pj = tex2D( tex_parts , k , cjd );
            #else
                pj = parts_j[ k ];
            #endif
            sort_j[k] = ( k << 16 ) | 
                (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj.x)*shiftn[0] + (shift[1]+pj.y)*shiftn[1] + (shift[2]+pj.z)*shiftn[2]) );
            }
            
        /* Make sure all the memory is in the right place. */
        // __threadfence_block();
        
        /* Sort using normalized bitonic sort. */
        cuda_sort_descending( sort_i , count_i );
        cuda_sort_ascending( sort_j , count_j );

        /* Store the sorted list back to global memory. */
        cuda_memcpy( sortlist , sort_i , sizeof(int) * count_i );
        cuda_memcpy( &sortlist[count_i] , sort_j , sizeof(int) * count_j );
            
        } /* re-build sorted pairs list. */
        
    /* Otherwise, just read it from memory. */
    else {
        cuda_memcpy( sort_i , sortlist , sizeof(int) * count_i );
        cuda_memcpy( sort_j , &sortlist[count_i] , sizeof(int) * count_j );
        // __threadfence_block();
        }
        
    TIMER_TOC2(tid_sort)
        
        
    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dmaxdist )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd = pjdid + threadID ) >= wrap )
                pjd -= wrap;
            
            /* Do we have a pair? */
            if ( pjd < cj ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                spjd = sort_j[pjd] >> 16;
                #ifdef PARTS_TEX
                    pj = tex2D( tex_parts , spjd , cjd );
                #else
                    pj = parts_j[ spjd ];
                #endif

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
                dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
                dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];
                    
                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pioff + (int)pj.w ] ) != 0 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    
                    /* Interact particles pi and pj. */
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pif[k] -= w;
                        forces_j[ 3*spjd + k ] += w;
                        }

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair4_verlet_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist , float *epot_global ) {
#else
__device__ void runner_dopair4_verlet_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist , float *epot_global ) {
#endif

    int k, pid, spid, pjdid, threadID, wrap, cj;
    int pioff;
    unsigned int dmaxdist;
    float4 pi, pj[4];
    int4 pot, pjd, spjd;
    char4 valid;
    float4 ee, eff, r2;
    float epot = 0.0f, w, nshift, inshift;
    float dx[12], pif[3], shift[3], shiftn[3];
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    /* if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        float *temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else */ {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;
       
    /* Re-build sorted pairs list? */
    if ( verlet_rebuild ) {
    
        TIMER_TIC2
        
        /* Pack the parts of i and j into the sort arrays. */
        for ( k = threadID ; k < count_i ; k += 4*cuda_frame ) {
            #ifdef PARTS_TEX
                pj[0] = tex2D( tex_parts , k + 0*cuda_frame , cid );
                pj[1] = tex2D( tex_parts , k + 1*cuda_frame , cid );
                pj[2] = tex2D( tex_parts , k + 2*cuda_frame , cid );
                pj[3] = tex2D( tex_parts , k + 3*cuda_frame , cid );
            #else
                pj[0] = parts_i[ k + 0*cuda_frame ];
                if ( k + 1*cuda_frame < count_i ) pj[1] = parts_i[ k + 1*cuda_frame ];
                if ( k + 2*cuda_frame < count_i ) pj[2] = parts_i[ k + 2*cuda_frame ];
                if ( k + 3*cuda_frame < count_i ) pj[3] = parts_i[ k + 3*cuda_frame ];
            #endif
            spjd.x = ( k << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[0].x*shiftn[0] + pj[0].y*shiftn[1] + pj[0].z*shiftn[2]) );
            spjd.y = ( (k + 1*cuda_frame) << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[1].x*shiftn[0] + pj[1].y*shiftn[1] + pj[1].z*shiftn[2]) );
            spjd.z = ( (k + 2*cuda_frame) << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[2].x*shiftn[0] + pj[2].y*shiftn[1] + pj[2].z*shiftn[2]) );
            spjd.w = ( (k + 3*cuda_frame) << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[3].x*shiftn[0] + pj[3].y*shiftn[1] + pj[3].z*shiftn[2]) );
            sort_i[k] = spjd.x;
            if ( k + 1*cuda_frame < count_i ) sort_i[ k + 1*cuda_frame ] = spjd.y;
            if ( k + 2*cuda_frame < count_i ) sort_i[ k + 2*cuda_frame ] = spjd.z;
            if ( k + 3*cuda_frame < count_i ) sort_i[ k + 3*cuda_frame ] = spjd.w;
            }
        for ( k = threadID ; k < count_j ; k += 4*cuda_frame ) {
            #ifdef PARTS_TEX
                pj[0] = tex2D( tex_parts , k + 0*cuda_frame , cjd );
                pj[1] = tex2D( tex_parts , k + 1*cuda_frame , cjd );
                pj[2] = tex2D( tex_parts , k + 2*cuda_frame , cjd );
                pj[3] = tex2D( tex_parts , k + 3*cuda_frame , cjd );
            #else
                pj[0] = parts_j[ k + 0*cuda_frame ];
                if ( k + 1*cuda_frame < count_j ) pj[1] = parts_j[ k + 1*cuda_frame ];
                if ( k + 2*cuda_frame < count_j ) pj[2] = parts_j[ k + 2*cuda_frame ];
                if ( k + 3*cuda_frame < count_j ) pj[3] = parts_j[ k + 3*cuda_frame ];
            #endif
            spjd.x = ( k << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[0].x)*shiftn[0] + (shift[1]+pj[0].y)*shiftn[1] + (shift[2]+pj[0].z)*shiftn[2]) );
            spjd.y = ( k + 1*cuda_frame << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[1].x)*shiftn[0] + (shift[1]+pj[1].y)*shiftn[1] + (shift[2]+pj[1].z)*shiftn[2]) );
            spjd.z = ( k + 2*cuda_frame << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[2].x)*shiftn[0] + (shift[1]+pj[2].y)*shiftn[1] + (shift[2]+pj[2].z)*shiftn[2]) );
            spjd.w = ( k + 3*cuda_frame << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[3].x)*shiftn[0] + (shift[1]+pj[3].y)*shiftn[1] + (shift[2]+pj[3].z)*shiftn[2]) );
            sort_j[k] = spjd.x;
            if ( k + 1*cuda_frame < count_j ) sort_j[ k + 1*cuda_frame ] = spjd.y;
            if ( k + 2*cuda_frame < count_j ) sort_j[ k + 2*cuda_frame ] = spjd.z;
            if ( k + 3*cuda_frame < count_j ) sort_j[ k + 3*cuda_frame ] = spjd.w;
            }
        
        TIMER_TOC2(tid_pack)
            
        /* Make sure all the memory is in the right place. */
        // __threadfence_block();
        
        /* Sort using normalized bitonic sort. */
        cuda_sort_descending( sort_i , count_i );
        cuda_sort_ascending( sort_j , count_j );

        /* Store the sorted list back to global memory. */
        cuda_memcpy( sortlist , sort_i , sizeof(int) * count_i );
        cuda_memcpy( &sortlist[count_i] , sort_j , sizeof(int) * count_j );
            
        } /* re-build sorted pairs list. */
        
    /* Otherwise, just read it from memory. */
    else {
        cuda_memcpy( sort_i , sortlist , sizeof(int) * count_i );
        cuda_memcpy( sort_j , &sortlist[count_i] , sizeof(int) * count_j );
        // __threadfence_block();
        }
        
        
    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dmaxdist )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd.x = pjdid + threadID ) >= wrap )
                pjd.x -= wrap;
            if ( ( pjd.y = pjdid + threadID + 1 ) >= wrap )
                pjd.y -= wrap;
            if ( ( pjd.z = pjdid + threadID + 2 ) >= wrap )
                pjd.z -= wrap;
            if ( ( pjd.w = pjdid + threadID + 3 ) >= wrap )
                pjd.w -= wrap;
                
            /* Get the particle pointers. */
            spjd.x = sort_j[pjd.x] >> 16; spjd.y = sort_j[pjd.y] >> 16; spjd.z = sort_j[pjd.z] >> 16; spjd.w = sort_j[pjd.w] >> 16; 
            #ifdef PARTS_TEX
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? tex2D( tex_parts , spjd.x , cjd ) : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? tex2D( tex_parts , spjd.y , cjd ) : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? tex2D( tex_parts , spjd.z , cjd ) : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? tex2D( tex_parts , spjd.w , cjd ) : pi;
            #else
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? parts_j[ spjd.x ] : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? parts_j[ spjd.y ] : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? parts_j[ spjd.z ] : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? parts_j[ spjd.w ] : pi;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pi.x - pj[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pi.y - pj[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pi.z - pj[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pi.x - pj[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pi.y - pj[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pi.z - pj[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pi.x - pj[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pi.y - pj[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pi.z - pj[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pi.x - pj[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pi.y - pj[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pi.z - pj[3].z; r2.w += dx[11] * dx[11];
                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? cuda_pind[ pioff + (int)pj[0].w ] : 0;
            pot.y = valid.y ? cuda_pind[ pioff + (int)pj[1].w ] : 0;
            pot.z = valid.z ? cuda_pind[ pioff + (int)pj[2].w ] : 0;
            pot.w = valid.w ? cuda_pind[ pioff + (int)pj[3].w ] : 0;
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Update the particle forces. */
            if ( valid.x ) {
                pif[0] -= ( w = eff.x * dx[0] ); forces_j[ 3*spjd.x + 0 ] += w;
                pif[1] -= ( w = eff.x * dx[1] ); forces_j[ 3*spjd.x + 1 ] += w;
                pif[2] -= ( w = eff.x * dx[2] ); forces_j[ 3*spjd.x + 2 ] += w;
                epot += ee.x;
                }
            // __threadfence_block();
            if ( valid.y ) {
                pif[0] -= ( w = eff.y * dx[3] ); forces_j[ 3*spjd.y + 0 ] += w;
                pif[1] -= ( w = eff.y * dx[4] ); forces_j[ 3*spjd.y + 1 ] += w;
                pif[2] -= ( w = eff.y * dx[5] ); forces_j[ 3*spjd.y + 2 ] += w;
                epot += ee.y;
                }
            // __threadfence_block();
            if ( valid.z ) {
                pif[0] -= ( w = eff.z * dx[6] ); forces_j[ 3*spjd.z + 0 ] += w;
                pif[1] -= ( w = eff.z * dx[7] ); forces_j[ 3*spjd.z + 1 ] += w;
                pif[2] -= ( w = eff.z * dx[8] ); forces_j[ 3*spjd.z + 2 ] += w;
                epot += ee.z;
                }
            // __threadfence_block();
            if ( valid.w ) {
                pif[0] -= ( w = eff.w * dx[9] ); forces_j[ 3*spjd.w + 0 ] += w;
                pif[1] -= ( w = eff.w * dx[10] ); forces_j[ 3*spjd.w + 1 ] += w;
                pif[2] -= ( w = eff.w * dx[11] ); forces_j[ 3*spjd.w + 2 ] += w;
                epot += ee.w;
                }
            // __threadfence_block();
            
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair_sorted_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , float *epot_global ) {
#else
__device__ void runner_dopair_sorted_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , float *epot_global ) {
#endif

    int k, pid, pjd, spid, spjd, pjdid, threadID, wrap, cj;
    int pioff, dcutoff;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, r2, w, ee = 0.0f, eff = 0.0f, nshift, inshift;
    float dx[3], pif[3], shift[3], shiftn[3];
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    /* if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        float *temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else */ {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dcutoff = 2 + cuda_dscale * cuda_cutoff;
       
    TIMER_TIC2
       
    /* Pack the parts of i and j into the sort arrays. */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , k , cid );
        #else
            pi = parts_i[ k ];
        #endif
        sort_i[k] = ( k << 16 ) |
            (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , k , cjd );
        #else
            pj = parts_j[ k ];
        #endif
        sort_j[k] = ( k << 16 ) | 
            (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj.x)*shiftn[0] + (shift[1]+pj.y)*shiftn[1] + (shift[2]+pj.z)*shiftn[2]) );
        }
        
    /* Make sure all the memory is in the right place. */
    // __threadfence_block();
    
    /* Sort using normalized bitonic sort. */
    cuda_sort_descending( sort_i , count_i );
    cuda_sort_ascending( sort_j , count_j );

        
    TIMER_TOC2(tid_sort)
        

    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dcutoff )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd = pjdid + threadID ) >= wrap )
                pjd -= wrap;
            
            /* Do we have a pair? */
            if ( pjd < cj ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                spjd = sort_j[pjd] >> 16;
                #ifdef PARTS_TEX
                    pj = tex2D( tex_parts , spjd , cjd );
                #else
                    pj = parts_j[ spjd ];
                #endif

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
                dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
                dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];
                    
                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pioff + (int)pj.w ] ) != 0 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    
                    /* Interact particles pi and pj. */
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pif[k] -= w;
                        forces_j[ 3*spjd + k ] += w;
                        }

                    /* Sync the shared memory values. */
                    // __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair4_sorted_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , float *epot_global ) {
#else
__device__ void runner_dopair4_sorted_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , float *epot_global ) {
#endif

    int k, pid, spid, pjdid, threadID, wrap, cj;
    int pioff, dcutoff;
    float4 pi, pj[4];
    int4 pot, pjd, spjd;
    char4 valid;
    float4 ee, eff, r2;
    float epot = 0.0f, w, nshift, inshift;
    float dx[12], pif[3], shift[3], shiftn[3];
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    /* if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        float *temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else */ {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dcutoff = 2 + cuda_dscale * cuda_cutoff;
       
    TIMER_TIC2
       
    /* Pack the parts of i and j into the sort arrays. */
    /* for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , k , cid );
        #else
            pi = parts_i[ k ];
        #endif
        sort_i[k] = ( k << 16 ) |
            (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , k , cjd );
        #else
            pi = parts_j[ k ];
        #endif
        sort_j[k] = ( k << 16 ) | 
            (unsigned int)( cuda_dscale * (nshift + (shift[0]+pi.x)*shiftn[0] + (shift[1]+pi.y)*shiftn[1] + (shift[2]+pi.z)*shiftn[2]) );
        } */
        
    /* Pack the parts of i and j into the sort arrays. */
    for ( k = threadID ; k < count_i ; k += 4*cuda_frame ) {
        #ifdef PARTS_TEX
            pj[0] = tex2D( tex_parts , k + 0*cuda_frame , cid );
            pj[1] = tex2D( tex_parts , k + 1*cuda_frame , cid );
            pj[2] = tex2D( tex_parts , k + 2*cuda_frame , cid );
            pj[3] = tex2D( tex_parts , k + 3*cuda_frame , cid );
        #else
            pj[0] = parts_i[ k + 0*cuda_frame ];
            if ( k + 1*cuda_frame < count_i ) pj[1] = parts_i[ k + 1*cuda_frame ];
            if ( k + 2*cuda_frame < count_i ) pj[2] = parts_i[ k + 2*cuda_frame ];
            if ( k + 3*cuda_frame < count_i ) pj[3] = parts_i[ k + 3*cuda_frame ];
        #endif
        spjd.x = ( k << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[0].x*shiftn[0] + pj[0].y*shiftn[1] + pj[0].z*shiftn[2]) );
        spjd.y = ( (k + 1*cuda_frame) << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[1].x*shiftn[0] + pj[1].y*shiftn[1] + pj[1].z*shiftn[2]) );
        spjd.z = ( (k + 2*cuda_frame) << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[2].x*shiftn[0] + pj[2].y*shiftn[1] + pj[2].z*shiftn[2]) );
        spjd.w = ( (k + 3*cuda_frame) << 16 ) | (unsigned int)( cuda_dscale * (nshift + pj[3].x*shiftn[0] + pj[3].y*shiftn[1] + pj[3].z*shiftn[2]) );
        sort_i[k] = spjd.x;
        if ( k + 1*cuda_frame < count_i ) sort_i[ k + 1*cuda_frame ] = spjd.y;
        if ( k + 2*cuda_frame < count_i ) sort_i[ k + 2*cuda_frame ] = spjd.z;
        if ( k + 3*cuda_frame < count_i ) sort_i[ k + 3*cuda_frame ] = spjd.w;
        }
    for ( k = threadID ; k < count_j ; k += 4*cuda_frame ) {
        #ifdef PARTS_TEX
            pj[0] = tex2D( tex_parts , k + 0*cuda_frame , cjd );
            pj[1] = tex2D( tex_parts , k + 1*cuda_frame , cjd );
            pj[2] = tex2D( tex_parts , k + 2*cuda_frame , cjd );
            pj[3] = tex2D( tex_parts , k + 3*cuda_frame , cjd );
        #else
            pj[0] = parts_j[ k + 0*cuda_frame ];
            if ( k + 1*cuda_frame < count_j ) pj[1] = parts_j[ k + 1*cuda_frame ];
            if ( k + 2*cuda_frame < count_j ) pj[2] = parts_j[ k + 2*cuda_frame ];
            if ( k + 3*cuda_frame < count_j ) pj[3] = parts_j[ k + 3*cuda_frame ];
        #endif
        spjd.x = ( k << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[0].x)*shiftn[0] + (shift[1]+pj[0].y)*shiftn[1] + (shift[2]+pj[0].z)*shiftn[2]) );
        spjd.y = ( k + 1*cuda_frame << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[1].x)*shiftn[0] + (shift[1]+pj[1].y)*shiftn[1] + (shift[2]+pj[1].z)*shiftn[2]) );
        spjd.z = ( k + 2*cuda_frame << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[2].x)*shiftn[0] + (shift[1]+pj[2].y)*shiftn[1] + (shift[2]+pj[2].z)*shiftn[2]) );
        spjd.w = ( k + 3*cuda_frame << 16 ) | (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj[3].x)*shiftn[0] + (shift[1]+pj[3].y)*shiftn[1] + (shift[2]+pj[3].z)*shiftn[2]) );
        sort_j[k] = spjd.x;
        if ( k + 1*cuda_frame < count_j ) sort_j[ k + 1*cuda_frame ] = spjd.y;
        if ( k + 2*cuda_frame < count_j ) sort_j[ k + 2*cuda_frame ] = spjd.z;
        if ( k + 3*cuda_frame < count_j ) sort_j[ k + 3*cuda_frame ] = spjd.w;
        }
        
    /* Make sure all the memory is in the right place. */
    // __threadfence_block();
    
    TIMER_TOC2(tid_pack)
    
    /* Sort using normalized bitonic sort. */
    cuda_sort_descending( sort_i , count_i );
    cuda_sort_ascending( sort_j , count_j );
    

    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dcutoff )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd.x = pjdid + threadID ) >= wrap )
                pjd.x -= wrap;
            if ( ( pjd.y = pjdid + threadID + 1 ) >= wrap )
                pjd.y -= wrap;
            if ( ( pjd.z = pjdid + threadID + 2 ) >= wrap )
                pjd.z -= wrap;
            if ( ( pjd.w = pjdid + threadID + 3 ) >= wrap )
                pjd.w -= wrap;
                
            /* Get the particle pointers. */
            spjd.x = sort_j[pjd.x] >> 16; spjd.y = sort_j[pjd.y] >> 16; spjd.z = sort_j[pjd.z] >> 16; spjd.w = sort_j[pjd.w] >> 16; 
            #ifdef PARTS_TEX
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? tex2D( tex_parts , spjd.x , cjd ) : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? tex2D( tex_parts , spjd.y , cjd ) : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? tex2D( tex_parts , spjd.z , cjd ) : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? tex2D( tex_parts , spjd.w , cjd ) : pi;
            #else
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? parts_j[ spjd.x ] : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? parts_j[ spjd.y ] : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? parts_j[ spjd.z ] : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? parts_j[ spjd.w ] : pi;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pi.x - pj[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pi.y - pj[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pi.z - pj[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pi.x - pj[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pi.y - pj[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pi.z - pj[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pi.x - pj[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pi.y - pj[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pi.z - pj[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pi.x - pj[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pi.y - pj[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pi.z - pj[3].z; r2.w += dx[11] * dx[11];
                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? cuda_pind[ pioff + (int)pj[0].w ] : 0;
            pot.y = valid.y ? cuda_pind[ pioff + (int)pj[1].w ] : 0;
            pot.z = valid.z ? cuda_pind[ pioff + (int)pj[2].w ] : 0;
            pot.w = valid.w ? cuda_pind[ pioff + (int)pj[3].w ] : 0;
            
            /* if ( pot.x != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.y != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.z != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.w != 0 )
                atomicAdd( &cuda_rcount , 1 ); */
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Update the particle forces. */
            if ( valid.x ) {
                pif[0] -= ( w = eff.x * dx[0] ); forces_j[ 3*spjd.x + 0 ] += w;
                pif[1] -= ( w = eff.x * dx[1] ); forces_j[ 3*spjd.x + 1 ] += w;
                pif[2] -= ( w = eff.x * dx[2] ); forces_j[ 3*spjd.x + 2 ] += w;
                epot += ee.x;
                }
            // __threadfence_block();
            if ( valid.y ) {
                pif[0] -= ( w = eff.y * dx[3] ); forces_j[ 3*spjd.y + 0 ] += w;
                pif[1] -= ( w = eff.y * dx[4] ); forces_j[ 3*spjd.y + 1 ] += w;
                pif[2] -= ( w = eff.y * dx[5] ); forces_j[ 3*spjd.y + 2 ] += w;
                epot += ee.y;
                }
            // __threadfence_block();
            if ( valid.z ) {
                pif[0] -= ( w = eff.z * dx[6] ); forces_j[ 3*spjd.z + 0 ] += w;
                pif[1] -= ( w = eff.z * dx[7] ); forces_j[ 3*spjd.z + 1 ] += w;
                pif[2] -= ( w = eff.z * dx[8] ); forces_j[ 3*spjd.z + 2 ] += w;
                epot += ee.z;
                }
            // __threadfence_block();
            if ( valid.w ) {
                pif[0] -= ( w = eff.w * dx[9] ); forces_j[ 3*spjd.w + 0 ] += w;
                pif[1] -= ( w = eff.w * dx[10] ); forces_j[ 3*spjd.w + 1 ] += w;
                pif[2] -= ( w = eff.w * dx[11] ); forces_j[ 3*spjd.w + 2 ] += w;
                epot += ee.w;
                }
            // __threadfence_block();
            
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        // __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_doself_cuda ( int cid , int count , float *forces , float *epot_global ) {
#else
__device__ void runner_doself_cuda ( float4 *parts , int count , float *forces , float *epot_global ) {
#endif

    int k, pid, pjd, threadID;
    int pjoff;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, dx[3], pjf[3], r2, w, ee, eff;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in the cell, frame-wise. */
    for ( pjd = threadID ; pjd < count-1 ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cid );
        #else
            pj = parts[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        pjf[0] = 0.0f; pjf[1] = 0.0f; pjf[2] = 0.0f;
            
        /* Loop over the particles in cell_i. */
        for ( pid = pjd+1 ; pid < count ; pid++ ) {
        
            /* Get a handle on the wrapped particle pid in cell_i. */
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , pid , cid );
            #else
                pi = parts[ pid ];
            #endif

            /* Compute the radius between pi and pj. */
            r2 = 0.0f;
            dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
            dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
            dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];

            /* Set the null potential if anything is bad. */
            if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ pjoff + (int)pi.w ] ) != 0 ) {

                /* Interact particles pi and pj. */
                potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                /* Store the interaction force and energy. */
                epot += ee;
                for ( k = 0 ; k < 3 ; k++ ) {
                    w = eff * dx[k];
                    forces[ 3*pid + k ] -= w;
                    pjf[k] += w;
                    }

                /* Sync the shared memory values. */
                // __threadfence_block();
            
                } /* in range and potential. */

            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        // __threadfence_block();

        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_self)
    
    }
    
    
/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_doself4_cuda ( int cid , int count , float *forces , float *epot_global ) {
#else
__device__ void runner_doself4_cuda ( float4 *parts , int count , float *forces , float *epot_global ) {
#endif

    int k, pjd, ind, threadID;
    int pjoff;
    float4 pi[4], pj;
    int4 pot, pid;
    char4 valid;
    float4 r2, ee, eff;
    float epot = 0.0f, dx[12], pjf[3], w;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Make sure everybody is in the same place. */
    // __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count-1 ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cid );
        #else
            pj = parts[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        for ( k = 0 ; k < 3 ; k++ )
            pjf[k] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = pjd+1 ; ind < count ; ind += 4 ) {
        
            /* Get the particle indices. */
            pid.x = ind + 0;
            pid.y = ind + 1;
            pid.z = ind + 2;
            pid.w = ind + 3;
                
            /* Get the particle pointers. */
            #ifdef PARTS_TEX
                pi[0] = ( valid.x = ( pid.x < count ) ) ? tex2D( tex_parts , pid.x , cid ) : pj;
                pi[1] = ( valid.y = ( pid.y < count ) ) ? tex2D( tex_parts , pid.y , cid ) : pj;
                pi[2] = ( valid.z = ( pid.z < count ) ) ? tex2D( tex_parts , pid.z , cid ) : pj;
                pi[3] = ( valid.w = ( pid.w < count ) ) ? tex2D( tex_parts , pid.w , cid ) : pj;
            #else
                pi[0] = ( valid.x = ( pid.x < count ) ) ? parts[ pid.x ] : pj;
                pi[1] = ( valid.y = ( pid.y < count ) ) ? parts[ pid.y ] : pj;
                pi[2] = ( valid.z = ( pid.z < count ) ) ? parts[ pid.z ] : pj;
                pi[3] = ( valid.w = ( pid.w < count ) ) ? parts[ pid.w ] : pj;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pj.x - pi[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pj.y - pi[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pj.z - pi[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pj.x - pi[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pj.y - pi[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pj.z - pi[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pj.x - pi[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pj.y - pi[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pj.z - pi[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pj.x - pi[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pj.y - pi[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pj.z - pi[3].z; r2.w += dx[11] * dx[11];

                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? cuda_pind[ pjoff + (int)pi[0].w ] : 0;
            pot.y = valid.y ? cuda_pind[ pjoff + (int)pi[1].w ] : 0;
            pot.z = valid.z ? cuda_pind[ pjoff + (int)pi[2].w ] : 0;
            pot.w = valid.w ? cuda_pind[ pjoff + (int)pi[3].w ] : 0;
            
            /* if ( pot.x != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.y != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.z != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.w != 0 )
                atomicAdd( &cuda_rcount , 1 ); */
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Update the forces and energy. */
            if ( valid.x ) {
                pjf[0] -= ( w = eff.x * dx[0] ); forces[ 3*pid.x + 0 ] += w;
                pjf[1] -= ( w = eff.x * dx[1] ); forces[ 3*pid.x + 1 ] += w;
                pjf[2] -= ( w = eff.x * dx[2] ); forces[ 3*pid.x + 2 ] += w;
                epot += ee.x;
                }
            // __threadfence_block();
            if ( valid.y ) {
                pjf[0] -= ( w = eff.y * dx[3] ); forces[ 3*pid.y + 0 ] += w;
                pjf[1] -= ( w = eff.y * dx[4] ); forces[ 3*pid.y + 1 ] += w;
                pjf[2] -= ( w = eff.y * dx[5] ); forces[ 3*pid.y + 2 ] += w;
                epot += ee.y;
                }
            // __threadfence_block();
            if ( valid.z ) {
                pjf[0] -= ( w = eff.z * dx[6] ); forces[ 3*pid.z + 0 ] += w;
                pjf[1] -= ( w = eff.z * dx[7] ); forces[ 3*pid.z + 1 ] += w;
                pjf[2] -= ( w = eff.z * dx[8] ); forces[ 3*pid.z + 2 ] += w;
                epot += ee.z;
                }
            // __threadfence_block();
            if ( valid.w ) {
                pjf[0] -= ( w = eff.w * dx[9] ); forces[ 3*pid.w + 0 ] += w;
                pjf[1] -= ( w = eff.w * dx[10] ); forces[ 3*pid.w + 1 ] += w;
                pjf[2] -= ( w = eff.w * dx[11] ); forces[ 3*pid.w + 2 ] += w;
                epot += ee.w;
                }
            // __threadfence_block();
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        // __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_doself_diag_cuda ( int cid , int count , float *forces , float *epot_global ) {
#else
__device__ void runner_doself_diag_cuda ( float4 *parts , int count , float *forces , float *epot_global ) {
#endif

    int diag, k, dind, dind2, diag_max, step, pid, pjd, threadID;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, dx[3], r2, w[3], ee, eff;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Step along the number of diagonal entries. */
    diag_max = count * (count - 1) / 2; step = 1;
    for ( diag = 0 ; diag < diag_max ; diag += step ) {
    
        /* is it time for this thread to step in? */
        if ( diag == threadID ) {
            step = diag;
            dind = diag; dind2 = dind*(dind + 1)/2;
            diag = (diag + 2) * (diag + 1) / 2 - 1;
            }
            
        /* If running, continue with the interactions. */
        if ( diag >= threadID && diag < diag_max ) {
        
            /* Increase the step if necessary. */
            if ( step < cuda_frame )
                step += 1;
    
            /* Get the location of the kth entry on the diagonal. */
            if ( diag >= dind2 + dind + 1 ) {
                dind += 1;
                dind2 += dind;
                }
            pjd = diag - dind2;
            pid = count - (dind - pjd) - 1;
            
            /* Get a handle on the particles. */
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , pid , cid );
                pj = tex2D( tex_parts , pjd , cid );
            #else
                pi = parts[ pid ];
                pj = parts[ pjd ];
            #endif

            /* Compute the radius between pi and pj. */
            r2 = 0.0f;
            dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
            dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
            dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];

            /* Set the null potential if anything is bad. */
            if ( r2 < cuda_cutoff2 && ( pot = cuda_pind[ ((int)pj.w)*cuda_maxtype + (int)pi.w ] ) != 0 ) {

                // atomicAdd( &cuda_rcount , 1 );
                    
                /* Interact particles pi and pj. */
                potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                /* Store the interaction force on pi and energy. */
                for ( k = 0 ; k < 3 ; k++ ) {
                    w[k] = eff * dx[k];
                    forces[ 3*pid + k ] -= w[k];
                    }

                /* Sync the shared memory values. */
                // __threadfence_block();

                /* Store the interaction force on pj. */
                epot += ee;
                for ( k = 0 ; k < 3 ; k++ )
                    forces[ 3*pjd + k ] += w[k];

                /* Sync the shared memory values. */
                // __threadfence_block();

                } /* range and potential? */

            /* printf( "runner_doself_diag_cuda[%i]: diag=%i, step=%i, i=%i, j=%i.\n" ,
                threadID , diag , step , pid , pjd ); */

            } /* is it this thread's turn? */
    
        } /* Loop over diagonal indices. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_self)
    
    }
    
    
/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_doself4_diag_cuda ( int cid , int count , float *forces , float *epot_global ) {
#else
__device__ void runner_doself4_diag_cuda ( float4 *parts , int count , float *forces , float *epot_global ) {
#endif

    int diag, k, diag_max, step, threadID, dind, dind2;
    float4 pi[4], pj[4], r2, ee, eff, w[3];
    int4 pot, ldiag, pid, pjd;
    char4 valid;
    float epot = 0.0f, dx[12];
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Step along the number of diagonal entries. */
    diag_max = count * (count - 1) / 2;
    step = 1; diag = 0;
    while ( diag < diag_max ) {
    
        /* Re-set the particle IDs. */
        pid = make_int4( 0 );
        pjd = make_int4( 0 );
    
        /* Get the diagonals and validity for each 4-step. */
        ldiag.x = diag;
        if ( ldiag.x == threadID ) {
            step = ldiag.x;
            dind = ldiag.x; dind2 = dind*(dind + 1)/2;
            ldiag.x = (diag + 2) * (diag + 1) / 2 - 1;
            }
        if ( ( valid.x = ( ldiag.x >= threadID && ldiag.x < diag_max ) ) ) {
            if ( step < cuda_frame )
                step += 1;
            if ( ldiag.x >= dind2 + dind + 1 ) {
                dind += 1;
                dind2 += dind;
                }
            pjd.x = ldiag.x - dind2;
            pid.x = count - (dind - pjd.x) - 1;
            }
        ldiag.y = ldiag.x + step;
        if ( ldiag.y == threadID ) {
            step = ldiag.y;
            dind = ldiag.y; dind2 = dind*(dind + 1)/2;
            ldiag.y = (ldiag.y + 2) * (ldiag.y + 1) / 2 - 1;
            }
        if ( ( valid.y = ( ldiag.y >= threadID && ldiag.y < diag_max ) ) ) {
            if ( step < cuda_frame )
                step += 1;               
            if ( ldiag.y >= dind2 + dind + 1 ) {
                dind += 1;
                dind2 += dind;
                }
            pjd.y = ldiag.y - dind2;
            pid.y = count - (dind - pjd.y) - 1;
            }
        ldiag.z = ldiag.y + step;
        if ( ldiag.z == threadID ) {
            step = ldiag.z;
            dind = ldiag.z; dind2 = dind*(dind + 1)/2;
            ldiag.z = (ldiag.z + 2) * (ldiag.z + 1) / 2 - 1;
            }
        if ( ( valid.z = ( ldiag.z >= threadID && ldiag.z < diag_max ) ) ) {
            if ( step < cuda_frame )
                step += 1;
            if ( ldiag.z >= dind2 + dind + 1 ) {
                dind += 1;
                dind2 += dind;
                }
            pjd.z = ldiag.z - dind2;
            pid.z = count - (dind - pjd.z) - 1;
            }
        ldiag.w = ldiag.z + step;
        if ( ldiag.w == threadID ) {
            step = ldiag.w;
            dind = ldiag.w; dind2 = dind*(dind + 1)/2;
            ldiag.w = (ldiag.w + 2) * (ldiag.w + 1) / 2 - 1;
            }
        if ( ( valid.w = ( ldiag.w >= threadID && ldiag.w < diag_max ) ) ) {
            if ( step < cuda_frame )
                step += 1;
            if ( ldiag.w >= dind2 + dind + 1 ) {
                dind += 1;
                dind2 += dind;
                }
            pjd.w = ldiag.w - dind2;
            pid.w = count - (dind - pjd.w) - 1;
            }
                
        /* Update diag for next iteration. */
        diag = ldiag.w + step;
    
        /* Get a handle on the particles. */
        #ifdef PARTS_TEX
            pi[0] = tex2D( tex_parts , pid.x , cid ); pj[0] = tex2D( tex_parts , pjd.x , cid );
            pi[1] = tex2D( tex_parts , pid.y , cid ); pj[1] = tex2D( tex_parts , pjd.y , cid );
            pi[2] = tex2D( tex_parts , pid.z , cid ); pj[2] = tex2D( tex_parts , pjd.z , cid );
            pi[3] = tex2D( tex_parts , pid.w , cid ); pj[3] = tex2D( tex_parts , pjd.w , cid );
        #else
            pi[0] = parts[ pid.x ]; pj[0] = parts[ pjd.x ];
            pi[1] = parts[ pid.y ]; pj[1] = parts[ pjd.y ];
            pi[2] = parts[ pid.z ]; pj[2] = parts[ pjd.z ];
            pi[3] = parts[ pid.w ]; pj[3] = parts[ pjd.w ];
        #endif

        /* Compute the radius between pi and pj. */
        r2 = make_float4( 0.0f );
        dx[0] = pi[0].x - pj[0].x; r2.x += dx[0]*dx[0];
        dx[1] = pi[0].y - pj[0].y; r2.x += dx[1]*dx[1];
        dx[2] = pi[0].z - pj[0].z; r2.x += dx[2]*dx[2];
        dx[3] = pi[1].x - pj[1].x; r2.y += dx[3]*dx[3];
        dx[4] = pi[1].y - pj[1].y; r2.y += dx[4]*dx[4];
        dx[5] = pi[1].z - pj[1].z; r2.y += dx[5]*dx[5];
        dx[6] = pi[2].x - pj[2].x; r2.z += dx[6]*dx[6];
        dx[7] = pi[2].y - pj[2].y; r2.z += dx[7]*dx[7];
        dx[8] = pi[2].z - pj[2].z; r2.z += dx[8]*dx[8];
        dx[9] = pi[3].x - pj[3].x; r2.w += dx[9]*dx[9];
        dx[10] = pi[3].y - pj[3].y; r2.w += dx[10]*dx[10];
        dx[11] = pi[3].z - pj[3].z; r2.w += dx[11]*dx[11];
        
        /* Get the potential for each pair. */
        pot.x = ( valid.x = ( valid.x && r2.x < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[0].w)*cuda_maxtype + (int)pi[0].w ] : 0;
        pot.y = ( valid.y = ( valid.y && r2.y < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[1].w)*cuda_maxtype + (int)pi[1].w ] : 0;
        pot.z = ( valid.z = ( valid.z && r2.z < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[2].w)*cuda_maxtype + (int)pi[2].w ] : 0;
        pot.w = ( valid.w = ( valid.w && r2.w < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[3].w)*cuda_maxtype + (int)pi[3].w ] : 0;

        /* Interact particles pi and pj. */
        potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
        
        /* Store the interaction force on pi and energy. */
        if ( valid.x ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.x + k ] -= ( w[k].x = eff.x * dx[k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.x + k ] += w[k].x;
            epot += ee.x;
            }
        // __threadfence_block();
        if ( valid.y ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.y + k ] -= ( w[k].y = eff.y * dx[3+k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.y + k ] += w[k].y;
            epot += ee.y;
            }
        // __threadfence_block();
         if ( valid.z ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.z + k ] -= ( w[k].z = eff.z * dx[6+k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.z + k ] += w[k].z;
            epot += ee.z;
            }
        // __threadfence_block();
        if ( valid.w ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.w + k ] -= ( w[k].w = eff.w * dx[9+k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.w + k ] += w[k].w;
            epot += ee.w;
            }
        // __threadfence_block();

        } /* Loop over diagonal indices. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_self)
    
    }
    

#ifdef PARTS_TEX
__device__ void runner_doself4_diag_cuda_old ( int cid , int count , float *forces , float *epot_global ) {
#else
__device__ void runner_doself4_diag_cuda_old ( float4 *parts , int count , float *forces , float *epot_global ) {
#endif

    int diag, k, diag_max, step, threadID;
    uint4 packed;
    float4 pi[4], pj[4], r2, ee, eff, w[3];
    int4 pot, ldiag, pid, pjd;
    char4 valid;
    float epot = 0.0f, dx[12];
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Step along the number of diagonal entries. */
    diag_max = count * (count - 1) / 2; step = 1; diag = 0;
    while ( diag < diag_max ) {
    
        /* Get the diagonals and validity for each 4-step. */
        ldiag.x = diag;
        if ( ldiag.x == threadID ) {
            step = ldiag.x;
            ldiag.x = (diag + 2) * (diag + 1) / 2 - 1;
            }
        if ( ( valid.x = ( ldiag.x >= threadID && ldiag.x < diag_max ) ) )
            if ( step < cuda_frame )
                step += 1;
        ldiag.y = ldiag.x + step;
        if ( ldiag.y == threadID ) {
            step = ldiag.y;
            ldiag.y = (ldiag.y + 2) * (ldiag.y + 1) / 2 - 1;
            }
        if ( ( valid.y = ( ldiag.y >= threadID && ldiag.y < diag_max ) ) )
            if ( step < cuda_frame )
                step += 1;               
        ldiag.z = ldiag.y + step;
        if ( ldiag.z == threadID ) {
            step = ldiag.z;
            ldiag.z = (ldiag.z + 2) * (ldiag.z + 1) / 2 - 1;
            }
        if ( ( valid.z = ( ldiag.z >= threadID && ldiag.z < diag_max ) ) )
            if ( step < cuda_frame )
                step += 1;
        ldiag.w = ldiag.z + step;
        if ( ldiag.w == threadID ) {
            step = ldiag.w;
            ldiag.w = (ldiag.w + 2) * (ldiag.w + 1) / 2 - 1;
            }
        if ( ( valid.w = ( ldiag.w >= threadID && ldiag.w < diag_max ) ) )
            if ( step < cuda_frame )
                step += 1;
                
        /* Update diag for next iteration. */
        diag = ldiag.w + step;
    
        /* Get the location of the kth entry on the diagonal. */
        packed.x = ( valid.x ) ? cuda_diags[ ldiag.x ] : 0;
        packed.y = ( valid.y ) ? cuda_diags[ ldiag.y ] : 0;
        packed.z = ( valid.z ) ? cuda_diags[ ldiag.z ] : 0;
        packed.w = ( valid.w ) ? cuda_diags[ ldiag.w ] : 0;
        pid.x = packed.x >> 16; pjd.x = count - (packed.x & 0xffff);
        pid.y = packed.y >> 16; pjd.y = count - (packed.y & 0xffff);
        pid.z = packed.z >> 16; pjd.z = count - (packed.z & 0xffff);
        pid.w = packed.w >> 16; pjd.w = count - (packed.w & 0xffff);
        
        /* Get a handle on the particles. */
        #ifdef PARTS_TEX
            pi[0] = tex2D( tex_parts , pid.x , cid ); pj[0] = tex2D( tex_parts , pjd.x , cid );
            pi[1] = tex2D( tex_parts , pid.y , cid ); pj[1] = tex2D( tex_parts , pjd.y , cid );
            pi[2] = tex2D( tex_parts , pid.z , cid ); pj[2] = tex2D( tex_parts , pjd.z , cid );
            pi[3] = tex2D( tex_parts , pid.w , cid ); pj[3] = tex2D( tex_parts , pjd.w , cid );
        #else
            pi[0] = parts[ pid.x ]; pj[0] = parts[ pjd.x ];
            pi[1] = parts[ pid.y ]; pj[1] = parts[ pjd.y ];
            pi[2] = parts[ pid.z ]; pj[2] = parts[ pjd.z ];
            pi[3] = parts[ pid.w ]; pj[3] = parts[ pjd.w ];
        #endif

        /* Compute the radius between pi and pj. */
        r2 = make_float4( 0.0f );
        dx[0] = pi[0].x - pj[0].x; r2.x += dx[0]*dx[0];
        dx[1] = pi[0].y - pj[0].y; r2.x += dx[1]*dx[1];
        dx[2] = pi[0].z - pj[0].z; r2.x += dx[2]*dx[2];
        dx[3] = pi[1].x - pj[1].x; r2.y += dx[3]*dx[3];
        dx[4] = pi[1].y - pj[1].y; r2.y += dx[4]*dx[4];
        dx[5] = pi[1].z - pj[1].z; r2.y += dx[5]*dx[5];
        dx[6] = pi[2].x - pj[2].x; r2.z += dx[6]*dx[6];
        dx[7] = pi[2].y - pj[2].y; r2.z += dx[7]*dx[7];
        dx[8] = pi[2].z - pj[2].z; r2.z += dx[8]*dx[8];
        dx[9] = pi[3].x - pj[3].x; r2.w += dx[9]*dx[9];
        dx[10] = pi[3].y - pj[3].y; r2.w += dx[10]*dx[10];
        dx[11] = pi[3].z - pj[3].z; r2.w += dx[11]*dx[11];
        
        /* Get the potential for each pair. */
        pot.x = ( valid.x = ( valid.x && r2.x < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[0].w)*cuda_maxtype + (int)pi[0].w ] : 0;
        pot.y = ( valid.y = ( valid.y && r2.y < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[1].w)*cuda_maxtype + (int)pi[1].w ] : 0;
        pot.z = ( valid.z = ( valid.z && r2.z < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[2].w)*cuda_maxtype + (int)pi[2].w ] : 0;
        pot.w = ( valid.w = ( valid.w && r2.w < cuda_cutoff2) ) ? cuda_pind[ ((int)pj[3].w)*cuda_maxtype + (int)pi[3].w ] : 0;

        /* Interact particles pi and pj. */
        potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
        
        /* Store the interaction force on pi and energy. */
        if ( valid.x ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.x + k ] -= ( w[k].x = eff.x * dx[k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.x + k ] += w[k].x;
            epot += ee.x;
            }
        // __threadfence_block();
        if ( valid.y ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.y + k ] -= ( w[k].y = eff.y * dx[3+k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.y + k ] += w[k].y;
            epot += ee.y;
            }
        // __threadfence_block();
         if ( valid.z ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.z + k ] -= ( w[k].z = eff.z * dx[6+k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.z + k ] += w[k].z;
            epot += ee.z;
            }
        // __threadfence_block();
        if ( valid.w ) {
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pid.w + k ] -= ( w[k].w = eff.w * dx[9+k] );
            // __threadfence_block();
            for ( k = 0 ; k < 3 ; k++ )
                forces[ 3*pjd.w + k ] += w[k].w;
            epot += ee.w;
            }
        // __threadfence_block();

        } /* Loop over diagonal indices. */
        
    /* Store the potential energy. */
    *epot_global += epot;
        
    TIMER_TOC(tid_self)
    
    }
    

/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus not
 * be externalized.
 */
 
int runner_bind ( hipArray *cuArray_coeffs , hipArray *cuArray_pind , hipArray *cuArray_diags ) {

    /* Set the coeff properties. */
    tex_coeffs.addressMode[0] = hipAddressModeClamp;
    tex_coeffs.addressMode[1] = hipAddressModeClamp;
    tex_coeffs.filterMode = hipFilterModePoint;
    tex_coeffs.normalized = false;

    /* Bind the coeffs. */
    cuda_coeffs = cuArray_coeffs;
    if ( hipBindTextureToArray( tex_coeffs , cuArray_coeffs ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Set the coeff properties. */
    tex_pind.addressMode[0] = hipAddressModeClamp;
    tex_pind.filterMode = hipFilterModePoint;
    tex_pind.normalized = false;

    /* Bind the pinds. */
    if ( hipBindTextureToArray( tex_pind , cuArray_pind ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Set the coeff properties. */
    tex_diags.addressMode[0] = hipAddressModeClamp;
    tex_diags.filterMode = hipFilterModePoint;
    tex_diags.normalized = false;

    /* Bind the diags. */
    if ( hipBindTextureToArray( tex_diags , cuArray_diags ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Rock and roll. */
    return runner_err_ok;

    }


/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus not
 * be externalized.
 */
 
int runner_parts_bind ( hipArray *cuArray_parts ) {

    /* Set the texture properties. */
    tex_parts.addressMode[0] = hipAddressModeClamp;
    tex_parts.addressMode[1] = hipAddressModeClamp;
    tex_parts.filterMode = hipFilterModePoint;
    tex_parts.normalized = false;

    /* Bind the parts. */
    if ( hipBindTextureToArray( tex_parts , cuArray_parts ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Rock and roll. */
    return runner_err_ok;

    }


/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus not
 * be externalized.
 */
 
int runner_parts_unbind ( ) {

    /* Bind the coeffs. */
    if ( hipUnbindTexture( tex_parts ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Rock and roll. */
    return runner_err_ok;

    }


/** This set of defines and includes produces kernels with buffers for multiples
 *  of 32 particles up to 512 cuda_maxparts.
 */
 
#define cuda_nrparts 32
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 64
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 96
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 128
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 160
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 192
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 224
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 256
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 288
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 320
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 352
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 384
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 416
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 448
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 480
    #include "runner_cuda_main.h"
#undef cuda_nrparts

// #define cuda_nrparts 512
//     #include "runner_cuda_main.h"



/**
 * @brief Offload and compute the nonbonded interactions on a CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_nonbond_cuda ( struct engine *e ) {

    dim3 nr_threads( cuda_frame , 1 );
    dim3 nr_blocks( e->nr_runners , 1 );
    int maxcount;
    ticks tic;
    // int zero = 0;
    // int cuda_io[32];
    // float cuda_fio[32];
    #ifdef TIMERS
        float timers[ tid_count ];
        double icpms = 1000.0 / 1.4e9; 
    #endif

    /* Load the particle data onto the device. */
    tic = getticks();
    if ( ( maxcount = engine_cuda_load_parts( e ) ) < 0 )
        return error(engine_err);
    e->timers[ engine_timer_cuda_load ] += getticks() - tic;

    /* Init the pointer to the next entry. */    
    /* if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_pair_next" ), &zero , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_tuple_next" ), &zero , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda); */
    /* if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_rcount" ), &zero , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda); */
    /* if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_pair_curr" ), &zero , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda); */
    /* if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_pairs_done" ), &zero , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda); */
    /* if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_cell_mutex" ), &zero , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda); */
        
    /* Re-set timers */
    #ifdef TIMERS
        for ( int k = 0 ; k < tid_count ; k++ )
            timers[k] = 0.0f;
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_timers ), timers , sizeof(float) * tid_count , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    #endif
    
    /* Start the appropriate kernel. */
    tic = getticks();
    if ( e->flags & engine_flag_verlet )
        switch ( (maxcount + 31) / 32 ) {
            case 1:
                runner_run_verlet_cuda_32 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 2:
                runner_run_verlet_cuda_64 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 3:
                runner_run_verlet_cuda_96 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 4:
                runner_run_verlet_cuda_128 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 5:
                runner_run_verlet_cuda_160 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 6:
                runner_run_verlet_cuda_192 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 7:
                runner_run_verlet_cuda_224 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 8:
                runner_run_verlet_cuda_256 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 9:
                runner_run_verlet_cuda_288 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 10:
                runner_run_verlet_cuda_320 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 11:
                runner_run_verlet_cuda_352 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 12:
                runner_run_verlet_cuda_384 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 13:
                runner_run_verlet_cuda_416 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 14:
                runner_run_verlet_cuda_448 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            case 15:
                runner_run_verlet_cuda_480 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
                break;
            // case 16:
            //     runner_run_verlet_cuda_512 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda , e->s.verlet_rebuild );
            //     break;
            default:
                return error(engine_err_maxparts);
            }
    else
        switch ( (maxcount + 31) / 32 ) {
            case 1:
                runner_run_cuda_32 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 2:
                runner_run_cuda_64 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 3:
                runner_run_cuda_96 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 4:
                runner_run_cuda_128 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 5:
                runner_run_cuda_160 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 6:
                runner_run_cuda_192 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 7:
                runner_run_cuda_224 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 8:
                runner_run_cuda_256 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 9:
                runner_run_cuda_288 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 10:
                runner_run_cuda_320 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 11:
                runner_run_cuda_352 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 12:
                runner_run_cuda_384 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 13:
                runner_run_cuda_416 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 14:
                runner_run_cuda_448 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            case 15:
                runner_run_cuda_480 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
                break;
            // case 16:
            //     runner_run_cuda_512 <<<nr_blocks,nr_threads>>> ( e->s.forces_cuda , e->s.counts_cuda , e->s.ind_cuda );
            //     break;
            default:
                return error(engine_err_maxparts);
            }
    if ( hipDeviceSynchronize() != hipSuccess )
        return cuda_error(engine_err_cuda);
    e->timers[ engine_timer_cuda_dopairs ] += getticks() - tic;
    
    /* Check for CUDA errors. */
    if ( hipPeekAtLastError() != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Get and dump timers. */
    #ifdef TIMERS
        if ( hipMemcpyFromSymbol( timers , HIP_SYMBOL(cuda_timers ), sizeof(float) * tid_count , 0 , hipMemcpyDeviceToHost ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        printf( "engine_nonbond_cuda: timers = [ %.2f " , icpms * timers[0] );
        for ( int k = 1 ; k < tid_count ; k++ )
            printf( "%.2f " , icpms * timers[k] );
        printf( "] ms\n" );
    #endif
        
    /* Get the IO data. */
    /*if ( hipMemcpyFromSymbol( cuda_io , HIP_SYMBOL("cuda_io" ), sizeof(int) * 32 , 0 , hipMemcpyDeviceToHost ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyFromSymbol( cuda_fio , HIP_SYMBOL("cuda_fio" ), sizeof(float) * 32 , 0 , hipMemcpyDeviceToHost ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    printf( "engine_nonbond_cuda: cuda_io is [ %i , %i , %i , %i , %i , %i , %i , %i , %i , %i ].\n" , 
        cuda_io[0] , cuda_io[1] , cuda_io[2] , cuda_io[3] , cuda_io[4] , cuda_io[5] , cuda_io[6] , cuda_io[7] , cuda_io[8] , cuda_io[9] );
    printf( "engine_nonbond_cuda: cuda_fio is [ %f " , cuda_fio[0] );
    for ( int k = 1 ; k < 32 ; k++ )
        printf( ", %f " , cuda_fio[k] );
    printf( "]\n" ); */
    
    /* if ( hipMemcpyFromSymbol( &zero , HIP_SYMBOL("cuda_rcount" ), sizeof(int) , 0 , hipMemcpyDeviceToHost ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    printf( "engine_nonbond_cuda: computed %i pairs.\n" , zero ); */

    /* Unload the particle data from the device. */
    tic = getticks();
    if ( engine_cuda_unload_parts( e ) < 0 )
        return error(engine_err);
    e->timers[ engine_timer_cuda_unload ] += getticks() - tic;

    /* Go away. */
    return engine_err_ok;
    
    }



/**
 * @brief Load the cell data onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return The maximum number of parts per cell or < 0
 *      on error (see #engine_err).
 */
 
extern "C" int engine_cuda_load_parts ( struct engine *e ) {
    
    int k, cid, pid, maxcount = 0;
    struct part *p;
    float4 *parts_cuda, *buff;
    struct space *s = &e->s;
    int *sortlists_ind, sortlists_count;
    struct cellpair_cuda *cellpairs;
    FPTYPE maxdist = s->cutoff + 2*s->verlet_maxdx;
    hipChannelFormatDesc channelDesc_float4 = hipCreateChannelDesc<float4>();
    
    /* Clear the counts array. */
    bzero( s->counts_cuda_local , sizeof(int) * s->nr_cells );
    
    /* Load the counts. */
    for ( k = 0 ; k < s->nr_marked ; k++ )
        if ( ( s->counts_cuda_local[ s->cid_marked[k] ] = s->cells[ s->cid_marked[k] ].count ) > maxcount )
            maxcount = s->counts_cuda_local[ s->cid_marked[k] ];
            
    /* Raise maxcount to the next multiple of 32. */
    maxcount = ( maxcount + (cuda_frame - 1) ) & ~(cuda_frame - 1);
    // printf( "engine_cuda_load_parts: maxcount=%i.\n" , maxcount );

    /* Compute the indices. */
    s->ind_cuda_local[0] = 0;
    for ( k = 1 ; k < s->nr_cells ; k++ )
        s->ind_cuda_local[k] = s->ind_cuda_local[k-1] + s->counts_cuda_local[k-1];
        
    /* Are we using verlet lists? */
    if ( e->flags & engine_flag_verlet ) {
    
        /* Start by setting the maxdist on the device. */
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_maxdist ), &maxdist , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    
        /* Do we need to re-build the list? */
        if ( s->verlet_rebuild ) {
        
            /* Get a copy of the cellpairs from the device. */
            if ( ( cellpairs = (struct cellpair_cuda *)alloca( sizeof(struct cellpair_cuda) * s->nr_pairs ) ) == NULL )
                return engine_err_malloc;
            if ( hipMemcpy( cellpairs , s->pairs_cuda , sizeof(struct cellpair_cuda) * s->nr_pairs , hipMemcpyDeviceToHost ) != hipSuccess )
                return cuda_error(engine_err_cuda);
        
            /* Allocate and fill the sortlist. */
            if ( ( sortlists_ind = (int *)alloca( sizeof(int) * (s->nr_pairs + 1) ) ) == NULL )
                return error(engine_err_malloc);
            sortlists_ind[0] = 0;
            for ( k = 1 ; k <= s->nr_pairs ; k++ )
                if ( cellpairs[k-1].i != cellpairs[k-1].j )
                    sortlists_ind[k] = sortlists_ind[k-1] + s->counts_cuda_local[cellpairs[k-1].i] + s->counts_cuda_local[cellpairs[k-1].j];
                else
                    sortlists_ind[k] = sortlists_ind[k-1];
            sortlists_count = sortlists_ind[s->nr_pairs];
            
            /* Do we need to re-allocate the sortlists? */
            if ( e->sortlists_cuda == NULL || e->sortlists_size < sortlists_count ) {
                e->sortlists_size = sortlists_count * 1.2;
                if ( e->sortlists_cuda != NULL && hipFree( e->sortlists_cuda ) != hipSuccess )
                    return cuda_error(engine_err_cuda);
                if ( hipMalloc( &e->sortlists_cuda , sizeof(int) * e->sortlists_size ) != hipSuccess )
                    return cuda_error(engine_err_cuda);
                if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_sortlists ), &e->sortlists_cuda , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
                    return cuda_error(engine_err_cuda);
                }
        
            /* Copy the indices over to the device. */
            if ( hipMemcpy( e->sortlists_ind_cuda , sortlists_ind , sizeof(int) * s->nr_pairs , hipMemcpyHostToDevice ) != hipSuccess )
                return cuda_error(engine_err_cuda);
        
            }
    
        } /* are we using verlet lists? */
    
    /* Allocate the particle buffer. */
    #ifdef PARTS_TEX
        if ( ( parts_cuda = (float4 *)malloc( sizeof( float4 ) * s->nr_cells * maxcount ) ) == NULL )
            return error(engine_err_malloc);
    #else
        if ( ( parts_cuda = (float4 *)malloc( sizeof( float4 ) * s->nr_parts ) ) == NULL )
            return error(engine_err_malloc);
    #endif
    
    /* Loop over the marked cells. */
    for ( k = 0 ; k < s->nr_marked ; k++ ) {
    
        /* Get the cell id. */
        cid = s->cid_marked[k];
        
        /* Copy the particle data to the device. */
        #ifdef PARTS_TEX
            buff = (float4 *)&parts_cuda[ maxcount * cid ];
        #else
            buff = (float4 *)&parts_cuda[ s->ind_cuda_local[cid] ];
        #endif
        for ( pid = 0 ; pid < s->counts_cuda_local[cid] ; pid++ ) {
            p = &s->cells[cid].parts[pid];
            buff[ pid ].x = p->x[0];
            buff[ pid ].y = p->x[1];
            buff[ pid ].z = p->x[2];
            buff[ pid ].w = p->type;
            }
    
        }
        
    // printf( "engine_cuda_load_parts: packed %i cells with %i parts each (%i kB).\n" , s->nr_cells , maxcount , (sizeof(float4)*maxcount*s->nr_cells)/1024 );
        
    /* Copy the counts onto the device. */
    if ( hipMemcpy( s->counts_cuda , s->counts_cuda_local , sizeof(int) * s->nr_cells , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Copy the inds onto the device. */
    if ( hipMemcpy( s->ind_cuda , s->ind_cuda_local , sizeof(int) * s->nr_cells , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Bind the particle positions to a texture. */
    #ifdef PARTS_TEX
        if ( hipMallocArray( (hipArray **)&s->cuArray_parts , &channelDesc_float4 , maxcount , s->nr_cells ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToArray( (hipArray *)s->cuArray_parts , 0 , 0 , parts_cuda , sizeof(float4) * s->nr_cells * maxcount , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( runner_parts_bind( (hipArray *)s->cuArray_parts ) < 0 )
            return error(engine_err_runner);
    #else
        if ( hipMalloc( &s->parts_cuda , sizeof( float4 ) * s->nr_parts ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpy( s->parts_cuda , parts_cuda , sizeof(float4) * s->nr_parts , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_parts ), &s->parts_cuda , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    #endif
    free( parts_cuda );
        
    /* Finally, init the forces on the device. */
    if ( hipMalloc( &s->forces_cuda , sizeof( float ) * 3 * s->nr_parts ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemset( s->forces_cuda , 0 , sizeof( float ) * 3 * s->nr_parts ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Our work is done here. */
    return maxcount;

    }
    
    

/**
 * @brief Load the cell data from the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_cuda_unload_parts ( struct engine *e ) {
    
    int k, cid, pid;
    struct part *p;
    float *forces_cuda, *buff, epot;
    struct space *s = &e->s;
    
    /* Get the forces from the device. */
    if ( ( forces_cuda = (float *)malloc( sizeof(float) * 3 * s->nr_parts ) ) == NULL )
        return error(engine_err_malloc);
    if ( hipMemcpy( forces_cuda , s->forces_cuda , sizeof(float) * 3 * s->nr_parts , hipMemcpyDeviceToHost ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Get the potential energy. */
    if ( hipMemcpyFromSymbol( &epot , HIP_SYMBOL(cuda_epot_out ), sizeof(float) , 0 , hipMemcpyDeviceToHost ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    e->s.epot += epot;
                
    /* Loop over the marked cells. */
    for ( k = 0 ; k < s->nr_marked ; k++ ) {
    
        /* Get the cell id. */
        cid = s->cid_marked[k];
        
        /* Copy the particle data from the device. */
        buff = &forces_cuda[ 3*s->ind_cuda_local[cid] ];
        for ( pid = 0 ; pid < e->s.cells[cid].count ; pid++ ) {
            p = &e->s.cells[cid].parts[pid];
            p->f[0] += buff[ 3*pid ];
            p->f[1] += buff[ 3*pid + 1 ];
            p->f[2] += buff[ 3*pid + 2 ];
            }
            
        }

    /* Deallocate the parts array and counts array. */
    free( forces_cuda );
    if ( hipFree( e->s.forces_cuda ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Unbind and free the parts data. */
    #ifdef PARTS_TEX
        if ( runner_parts_unbind( ) < 0 )
            return error(engine_err_runner);
        if ( hipFreeArray( (hipArray *)e->s.cuArray_parts ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    #else
        if ( hipFree( e->s.parts_cuda ) != hipSuccess )
            return cuda_error(engine_err_cuda);
    #endif
        
    /* Our work is done here. */
    return engine_err_ok;

    }


/**
 * @brief Load the potentials and cell pairs onto the CUDA device.
 *
 * @param e The #engine.
 *
 * @return #engine_err_ok or < 0 on error (see #engine_err).
 */
 
extern "C" int engine_cuda_load ( struct engine *e ) {

    int i, j, k, nr_pots, nr_coeffs, max_coeffs = 0;
    int pind[ e->max_type * e->max_type ], *pind_cuda;
    struct potential *pots[ e->nr_types * (e->nr_types + 1) / 2 + 1 ];
    struct cellpair_cuda *pairs_cuda;
    float *finger, *coeffs_cuda;
    float cutoff = e->s.cutoff, cutoff2 = e->s.cutoff2, dscale; //, buff[ e->nr_types ];
    hipArray *cuArray_coeffs, *cuArray_pind, *cuArray_diags;
    hipChannelFormatDesc channelDesc_int = hipCreateChannelDesc<int>();
    hipChannelFormatDesc channelDesc_float = hipCreateChannelDesc<float>();
    hipChannelFormatDesc channelDesc_float4 = hipCreateChannelDesc<float4>();
    void *devptr;
    unsigned int *taboo_cuda, *pairIDs_cuda, pairIDs[ e->s.nr_pairs ], *diags, *diags_cuda;
    
    /* Init the null potential. */
    if ( ( pots[0] = (struct potential *)alloca( sizeof(struct potential) ) ) == NULL )
        return error(engine_err_malloc);
    pots[0]->alpha[0] = pots[0]->alpha[1] = pots[0]->alpha[2] = pots[0]->alpha[3] = 0.0f;
    pots[0]->a = 0.0; pots[0]->b = DBL_MAX;
    pots[0]->flags = potential_flag_none;
    pots[0]->n = 0;
    if ( ( pots[0]->c = (FPTYPE *)alloca( sizeof(float) * potential_chunk ) ) == NULL )
        return error(engine_err_malloc);
    bzero( pots[0]->c , sizeof(float) * potential_chunk );
    nr_pots = 1; nr_coeffs = 1;
    
    /* Start by identifying the unique potentials in the engine. */
    for ( i = 0 ; i < e->max_type * e->max_type ; i++ ) {
    
        /* Skip if there is no potential or no parts of this type. */
        if ( e->p[i] == NULL )
            continue;
            
        /* Check this potential against previous potentials. */
        for ( j = 0 ; j < nr_pots && e->p[i] != pots[j] ; j++ );
        if ( j < nr_pots )
            continue;
            
        /* Store this potential and the number of coefficient entries it has. */
        pots[nr_pots] = e->p[i];
        nr_pots += 1;
        nr_coeffs += e->p[i]->n + 1;
        if ( e->p[i]->n + 1 > max_coeffs )
            max_coeffs = e->p[i]->n + 1;
    
        }
       
    /* Copy eps and rmin to the device. */
    /* for ( i = 0 ; i < e->nr_types ; i++ )
        buff[i] = sqrt( fabs( e->types[i].eps ) );
    if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_eps" ), buff , sizeof(float) * e->nr_types , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    for ( i = 0 ; i < e->nr_types ; i++ )
        buff[i] = e->types[i].rmin;
    if ( hipMemcpyToSymbol(HIP_SYMBOL( "cuda_rmin" ), buff , sizeof(float) * e->nr_types , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda); */

    /* Pack the potential matrix. */
    for ( i = 0 ; i < e->max_type * e->max_type ; i++ ) {
        if ( e->p[i] == NULL ) {
            pind[i] = 0;
            }
        else {
            for ( j = 0 ; j < nr_pots && pots[j] != e->p[i] ; j++ );
            pind[i] = j;
            }
        }
        
    /* Pack the coefficients before shipping them off to the device. */
    if ( ( coeffs_cuda = (float *)malloc( sizeof(float4) * (2*max_coeffs + 2) * nr_pots ) ) == NULL )
        return error(engine_err_malloc);
    for ( i = 0 ; i < nr_pots ; i++ ) {
        finger = &coeffs_cuda[ i*4*(2*max_coeffs + 2) ];
        finger[0] = pots[i]->alpha[0];
        finger[1] = pots[i]->alpha[1];
        finger[2] = pots[i]->alpha[2];
        memcpy( &finger[8] , pots[i]->c , sizeof(float) * potential_chunk * (pots[i]->n + 1) );
        }
    /* for ( finger = coeffs_cuda , i = 0 ; i < nr_pots ; i++ ) {
        memcpy( finger , pots[i]->c , sizeof(float) * potential_chunk * (pots[i]->n + 1) );
        finger = &finger[ (pots[i]->n + 1) * potential_chunk ];
        } */
    printf( "engine_cuda_load: packed %i potentials with %i coefficient chunks (%i kB).\n" , nr_pots , max_coeffs , (sizeof(float4)*(2*max_coeffs+2)*nr_pots)/1024 ); fflush(stdout);
        
    /* Bind the potential coefficients to a texture. */
    if ( hipMallocArray( &cuArray_coeffs , &channelDesc_float4 , 2*max_coeffs + 2 , nr_pots ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToArray( cuArray_coeffs , 0 , 0 , coeffs_cuda , sizeof(float4) * (2*max_coeffs + 2) * nr_pots , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    free( coeffs_cuda );
    
    /* Pack the diagonal offsets into a newly allocated array and 
       copy to the device. */
    if ( ( diags = (unsigned int *)alloca( sizeof(unsigned int) * cuda_ndiags ) ) == NULL )
        return error(engine_err_malloc);
    for ( i = 0 ; i < cuda_ndiags ; i++ ) {
        k = ( sqrt( 8.0*i + 1 ) - 1 ) / 2;
        diags[i] = ( ( i - k*(k+1)/2 ) << 16 ) | ( 1 + k - i + k*(k+1)/2 );
        }
    if ( hipMallocArray( &cuArray_diags , &channelDesc_int , cuda_ndiags , 1 ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToArray( cuArray_diags , 0 , 0 , diags , sizeof(int) * cuda_ndiags , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Store diags as a constant too. */
    if ( hipMalloc( &diags_cuda , sizeof(unsigned int) * cuda_ndiags ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpy( diags_cuda , diags , sizeof(unsigned int) * cuda_ndiags , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_diags ), &diags_cuda , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
            
    /* Copy the potential indices to the device. */
    if ( hipMallocArray( &cuArray_pind , &channelDesc_int , e->max_type * e->max_type , 1 ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToArray( cuArray_pind , 0 , 0 , pind , sizeof(int) * e->max_type * e->max_type , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Store pind as a constant too. */
    if ( hipMalloc( &pind_cuda , sizeof(unsigned int) * e->max_type * e->max_type ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpy( pind_cuda , pind , sizeof(unsigned int) * e->max_type * e->max_type , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_pind ), &pind_cuda , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
            
    /* Bind the textures on the device. */
    if ( runner_bind( cuArray_coeffs , cuArray_pind , cuArray_diags ) < 0 )
        return error(engine_err_runner);
        
        
    /* Set the constant pointer to the null potential and other useful values. */
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_cutoff2 ), &cutoff2 , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_cutoff ), &cutoff , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_maxdist ), &cutoff , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_maxtype ), &(e->max_type) , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    dscale = ((float)SHRT_MAX) / ( 3.0 * sqrt( e->s.h[0]*e->s.h[0] + e->s.h[1]*e->s.h[1] + e->s.h[2]*e->s.h[2] ) );
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_dscale ), &dscale , sizeof(float) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Allocate and fill the compact list of pairs. */
    if ( ( pairs_cuda = (struct cellpair_cuda *)alloca( sizeof(struct cellpair_cuda) * e->s.nr_pairs ) ) == NULL )
        return error(engine_err_malloc);
    for ( i = 0 ; i < e->s.nr_pairs ; i++ ) {
        pairs_cuda[i].i = e->s.pairs[i].i;
        pairs_cuda[i].j = e->s.pairs[i].j;
        pairs_cuda[i].shift[0] = e->s.pairs[i].shift[0];
        pairs_cuda[i].shift[1] = e->s.pairs[i].shift[1];
        pairs_cuda[i].shift[2] = e->s.pairs[i].shift[2];
        }
        
    /* Allocate and fill the pairs list on the device. */
    if ( hipMalloc( &e->s.pairs_cuda , sizeof(struct cellpair_cuda) * e->s.nr_pairs ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpy( e->s.pairs_cuda , pairs_cuda , sizeof(struct cellpair_cuda) * e->s.nr_pairs , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_pairs ), &(e->s.pairs_cuda) , sizeof(struct cellpair_cuda *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipGetSymbolAddress( &devptr , HIP_SYMBOL(cuda_fifos_in )) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemset( devptr , 0  , sizeof(struct fifo_cuda) * cuda_maxblocks ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipGetSymbolAddress( &devptr , HIP_SYMBOL(cuda_fifos_out )) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemset( devptr , 0  , sizeof(struct fifo_cuda) * cuda_maxblocks ) != hipSuccess )
        return cuda_error(engine_err_cuda);

        
    /* Allocate the sortlists locally and on the device if needed. */
    if ( e->flags & engine_flag_verlet ) {
        e->sortlists_cuda = NULL;
        if ( hipMalloc( &e->sortlists_ind_cuda , sizeof(int) * e->s.nr_pairs ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_sortlists_ind ), &e->sortlists_ind_cuda , sizeof(void *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
            return cuda_error(engine_err_cuda);
        }

    /* Set the number of pairs and cells. */
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_nr_pairs ), &(e->s.nr_pairs) , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_nr_cells ), &(e->s.nr_cells) , sizeof(int) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Allocate the cell counts and offsets. */
    if ( ( e->s.counts_cuda_local = (int *)malloc( sizeof(int) * e->s.nr_cells ) ) == NULL ||
         ( e->s.ind_cuda_local = (int *)malloc( sizeof(int) * e->s.nr_cells ) ) == NULL )
        return error(engine_err_malloc);
    if ( hipMalloc( &e->s.counts_cuda , sizeof(int) * e->s.nr_cells ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMalloc( &e->s.ind_cuda , sizeof(int) * e->s.nr_cells ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Allocate and init the taboo list on the device. */
    if ( hipMalloc( &taboo_cuda , sizeof(int) * e->s.nr_cells ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemset( taboo_cuda , 0 , sizeof(int) * e->s.nr_cells ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_taboo ), &taboo_cuda , sizeof(int *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Pack and send the pairIDs. */
    for ( k = 0 ; k < e->s.nr_pairs ; k++ )
        pairIDs[k] = k;
    if ( hipMalloc( &pairIDs_cuda , sizeof(int) * e->s.nr_pairs ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpy( pairIDs_cuda , pairIDs , sizeof(int) * e->s.nr_pairs , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    if ( hipMemcpyToSymbol(HIP_SYMBOL( cuda_pairIDs ), &pairIDs_cuda , sizeof(int *) , 0 , hipMemcpyHostToDevice ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* He's done it! */
    return engine_err_ok;
    
    }
    
    




