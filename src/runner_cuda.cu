#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of mdcore.
 * Coypright (c) 2012 Pedro Gonnet (gonnet@maths.ox.ac.uk)
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * 
 ******************************************************************************/

/* Include configuration header */
#include "../config.h"

/* Include some standard header files */
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <limits.h>

/* Include headers for overloaded vector functions. */
#include "cutil_math.h"

/* Include some conditional headers. */
#ifdef HAVE_MPI
    #include <mpi.h>
#endif

/* Force single precision. */
#define FPTYPE_SINGLE 1

/* Disable vectorization for the nvcc compiler's sake. */
#undef __SSE__
#undef __SSE2__
#undef __ALTIVEC__
#undef __AVX__

/* Include local headers */
#include "cycle.h"
#include "errs.h"
#include "fptype.h"
#include "part.h"
#include "cell.h"
#include "fifo.h"
#include "space.h"
#include "potential.h"
#include "engine.h"
#include "runner.h"
#include "runner_cuda.h"


/* the error macro. */
#define cuda_error(id)			( engine_err = errs_register( id , hipGetErrorString(hipGetLastError()) , __LINE__ , __FUNCTION__ , __FILE__ ) )


/* The constant null potential. */
__constant__ struct potential *potential_null_cuda = NULL;

/* The number of cells and pairs. */
__constant__ int cuda_nr_pairs = 0;
__device__ int cuda_pairs_done = 0;
__constant__ int cuda_nr_tuples = 0;
__constant__ int cuda_nr_cells = 0;

/* The parts (non-texture access). */
__constant__ float4 *cuda_parts;

/* The mutex for accessing the cell pair list. */
__device__ int cuda_cell_mutex = 0;
__device__ int cuda_barrier = 0;

/* The list of cell pairs. */
__constant__ struct cellpair_cuda *cuda_pairs;
__device__ int *cuda_taboo;

/* The index of the next free cell pair. */
__device__ int cuda_pair_next = 0;

/* Indices for the "new" queue. */
__device__ int cuda_pair_count = 0;
__device__ int cuda_pair_curr = 0;
__device__ int *cuda_pairIDs;

/* Some constants. */
__constant__ float cuda_cutoff2 = 0.0f;
__constant__ float cuda_cutoff = 0.0f;
__constant__ float cuda_dscale = 0.0f;
__constant__ float cuda_maxdist = 0.0f;
__constant__ struct potential **cuda_p;
__constant__ int cuda_maxtype = 0;
__constant__ struct potential *cuda_pots;

/* Sortlists for the Verlet algorithm. */
__device__ unsigned int *cuda_sortlists = NULL;
__device__ int *cuda_sortlists_ind;

/* The potential coefficients, as a texture. */
texture< float4 , hipTextureType2D > tex_coeffs;
texture< float4 , hipTextureType1D > tex_alphas;
texture< float4 , hipTextureType2D > tex_parts;

/* Other textures. */
texture< int , hipTextureType1D > tex_pind;
texture< unsigned int , hipTextureType1D > tex_diags;

/* Arrays to hold the textures. */
hipArray *cuda_coeffs, *cuda_alphas, *cuda_offsets, *cuda_pind, *cuda_diags;

/* The potential parameters (hard-wired size for now). */
__constant__ float cuda_eps[ 100 ];
__constant__ float cuda_rmin[ 100 ];

/* The list of fifos to work with. */
__device__ struct fifo_cuda cuda_fifos_in[ cuda_maxblocks ];
__device__ struct fifo_cuda cuda_fifos_out[ cuda_maxblocks ];

/* Use a set of variables to communicate with the outside world. */
__device__ float cuda_fio[32];
__device__ int cuda_io[32];
__device__ int cuda_rcount = 0;

/* Timers. */
__device__ float cuda_timers[ tid_count ];


/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ void cuda_mutex_lock ( int *m ) {
    TIMER_TIC
    while ( atomicCAS( m , 0 , 1 ) != 0 );
    TIMER_TOC( tid_mutex )
    }


/**
 * @brief Lock a device mutex with an additional condition.
 *
 * @param m The mutex.
 * @param c the condition
 *
 * @return @c 1 if the mutex could be locked or zero if the condition @c c
 * was reached first.
 *
 * Loops until the mutex can be set or until @c *c is non-zero.
 * Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ int cuda_mutex_lock_cond ( int *m , int *c ) {
    TIMER_TIC
    while ( atomicCAS( c , 0 , 0 ) == 0 )
        if ( atomicCAS( m , 0 , 1 ) == 0 ) {
            TIMER_TOC( tid_mutex )
            return 1;
            }
    TIMER_TOC( tid_mutex )
    return 0;
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ void cuda_mutex_unlock ( int *m ) {
    atomicExch( m , 0 );
    }
    
    
/**
 * @brief Push an element onto a #fifo_cuda, blocking.
 *
 * @return The number of elements in the #fifo_cuda.
 */
 
__device__ inline int cuda_fifo_push ( struct fifo_cuda *f , unsigned int e ) {

    /* Wait for there to be space in the list. */
    while ( f->count == cuda_fifo_size );

    /* Put the element in the list. */
    atomicExch( &(f->data[ f->last ]) , e );
    
    /* Increase the "last" counter. */
    atomicExch( &f->last , (f->last + 1) % cuda_fifo_size );
    
    /* Increase the count. */
    atomicAdd( &f->count , 1 );
    
    /* Return the fifo size. */
    return f->count;

    }
    
    
/**
 * @brief Pop an element from a #fifo_cuda, blocking.
 *
 * @return The popped element.
 */
 
__device__ inline unsigned int cuda_fifo_pop ( struct fifo_cuda *f ) {

    /* Wait for there to be something in the fifo. */
    while ( f->count == 0 );

    unsigned int res = f->data[ f->first ];

    /* Increase the "fist" counter. */
    atomicExch( &f->first , (f->first + 1) % cuda_fifo_size );
    
    /* Decrease the count. */
    atomicSub( &f->count , 1 );
    
    /* Return the first element. */
    return res;

    }
    
    
/**
 * @brief Copy bulk memory in a strided way.
 *
 * @param dest Pointer to destination memory.
 * @param source Pointer to source memory.
 * @param count Number of bytes to copy, must be a multiple of sizeof(int).
 */
 
__device__ inline void cuda_memcpy ( void *dest , void *source , int count ) {

    int j, k, icount = count / sizeof(int) / cuda_frame / cuda_memcpy_chunk;
    int *idest = (int *)dest, *isource = (int *)source;
    int chunk[cuda_memcpy_chunk];
    int threadID = threadIdx.x;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 ; k < icount ; k += 1 ) {
        for ( j = 0 ; j < cuda_memcpy_chunk ; j++ )
            chunk[j] = isource[ (cuda_memcpy_chunk*k+j)*cuda_frame + threadID ];
        for ( j = 0 ; j < cuda_memcpy_chunk ; j++ )
            idest[ (cuda_memcpy_chunk*k+j)*cuda_frame + threadID ] = chunk[j];
        }
    for ( k = cuda_memcpy_chunk*cuda_frame*icount + threadID ; k < count/sizeof(int) ; k += cuda_frame )
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
    }
    
    
__device__ inline void cuda_memcpy_old ( void *dest , void *source , int count ) {

    int k;
    volatile int *idest = (int *)dest, *isource = (int *)source;
    
    TIMER_TIC
    
    /* Copy the data in chunks of sizeof(int). */
    for ( k = 0 + threadIdx.x ; k < count/sizeof(int) ; k += cuda_frame )
        idest[k] = isource[k];
        
    TIMER_TOC(tid_memcpy)
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated) using
 *      texture memory on the device.
 *
 * @param pid The index of the #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda_tex ( int pid , float r2 , float *e , float *f ) {

    int ind;
    float x, ee, eff, r, ir;
    float4 alpha, c1, c2;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    alpha = tex1D( tex_alphas , pid );
    if ( ( ind = alpha.x + r * ( alpha.y + r * alpha.z ) ) < 0 )
        ind = 0;
    ind += alpha.w;
    
    /* pre-load the coefficients. */
    c1 = tex2D( tex_coeffs , 0 , ind );
    c2 = tex2D( tex_coeffs , 1 , ind );
    
    /* adjust x to the interval */
    x = (r - c1.x) * c1.y;
    
    /* compute the potential and its derivative */
    eff = c1.z;
    ee = c1.z * x + c1.w;
    eff = eff * x + ee;
    ee = ee * x + c2.x;
    eff = eff * x + ee;
    ee = ee * x + c2.y;
    eff = eff * x + ee;
    ee = ee * x + c2.z;
    eff = eff * x + ee;
    ee = ee * x + c2.w;

    /* store the result */
    *e = ee; *f = eff * c1.y * ir;
        
    TIMER_TOC(tid_potential)
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated) using
 *      texture memory on the device.
 *
 * @param pid The index of the #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval4_cuda_tex ( int4 pid , float4 r2 , float4 *e , float4 *f ) {

    int k;
    int4 ind;
    float4 x, ee, eff, r, ir, t1[4], t2[4], c[potential_chunk], a[4];
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir.x = rsqrtf(r2.x);
    ir.y = rsqrtf(r2.y);
    ir.z = rsqrtf(r2.z);
    ir.w = rsqrtf(r2.w);
    r = r2*ir;
    
    /* compute the interval index */
    a[0] = tex1D( tex_alphas , pid.x );
    a[1] = tex1D( tex_alphas , pid.y );
    a[2] = tex1D( tex_alphas , pid.z );
    a[3] = tex1D( tex_alphas , pid.w );
    ind.x = a[0].w + max( 0 , (int)( a[0].x + r.x * ( a[0].y + r.x * a[0].z ) ) );
    ind.y = a[1].w + max( 0 , (int)( a[1].x + r.y * ( a[1].y + r.y * a[1].z ) ) );
    ind.z = a[2].w + max( 0 , (int)( a[2].x + r.z * ( a[2].y + r.z * a[2].z ) ) );
    ind.w = a[3].w + max( 0 , (int)( a[3].x + r.w * ( a[3].y + r.w * a[3].z ) ) );
    
    /* pre-load the coefficients. */
    t1[0] = tex2D( tex_coeffs , 0 , ind.x );
    t2[0] = tex2D( tex_coeffs , 1 , ind.x );
    t1[1] = tex2D( tex_coeffs , 0 , ind.y );
    t2[1] = tex2D( tex_coeffs , 1 , ind.y );
    t1[2] = tex2D( tex_coeffs , 0 , ind.z );
    t2[2] = tex2D( tex_coeffs , 1 , ind.z );
    t1[3] = tex2D( tex_coeffs , 0 , ind.w );
    t2[3] = tex2D( tex_coeffs , 1 , ind.w );
    c[0] = make_float4( t1[0].x , t1[1].x , t1[2].x , t1[3].x );
    c[1] = make_float4( t1[0].y , t1[1].y , t1[2].y , t1[3].y );
    c[2] = make_float4( t1[0].z , t1[1].z , t1[2].z , t1[3].z );
    c[3] = make_float4( t1[0].w , t1[1].w , t1[2].w , t1[3].w );
    c[4] = make_float4( t2[0].x , t2[1].x , t2[2].x , t2[3].x );
    c[5] = make_float4( t2[0].y , t2[1].y , t2[2].y , t2[3].y );
    c[6] = make_float4( t2[0].z , t2[1].z , t2[2].z , t2[3].z );
    c[7] = make_float4( t2[0].w , t2[1].w , t2[2].w , t2[3].w );
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    eff = c[2];
    ee = c[2] * x + c[3];
    #pragma unroll
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda ( struct potential *p , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, *c, ir, r;
    
    TIMER_TIC
    
    /* Get r for the right type. */
    ir = rsqrtf(r2);
    r = r2*ir;
    
    /* compute the interval index */
    ind = fmaxf( 0.0f , p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]) );
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    #pragma unroll
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] * ir;
        
    TIMER_TOC(tid_potential)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */

#ifdef PARTS_TEX 
__device__ void runner_dopair_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , float *pshift ) {
#else
__device__ void runner_dopair_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , float *pshift ) {
#endif

    int k, pid, pjd, ind, wrap_i, threadID;
    int pjoff;
    int pot;
    float epot = 0.0f, dx[3], pjf[3], shift[3], r2, w;
    float ee = 0.0f, eff = 0.0f, *temp;
    float4 pi, pj;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i < ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

    /* Get the wraps. */
    wrap_i = (count_i < cuda_frame) ? cuda_frame : count_i;
    
    /* Make sure everybody is in the same place. */
    __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cjd );
        #else
            pj = parts_j[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        pj.x += shift[0]; pj.y += shift[1]; pj.z += shift[2];
        pjf[0] = 0.0f; pjf[1] = 0.0f; pjf[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = 0 ; ind < wrap_i ; ind++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pid = ind + threadID ) >= wrap_i )
                pid -= wrap_i;
            if ( pid < count_i ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                #ifdef PARTS_TEX
                    pi = tex2D( tex_parts , pid , cid );
                #else
                    pi = parts_i[ pid ];
                #endif

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
                dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
                dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];

                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = tex1D( tex_pind , pjoff + pi.w ) ) != 0 ) {

                    // atomicAdd( &cuda_rcount , 1 );
                
                    /* Interact particles pi and pj. */
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        forces_i[ 3*pid + k ] -= w;
                        pjf[k] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();
                
                    } /* in range and potential. */

                } /* valid pid? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_j[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */

#ifdef PARTS_TEX 
__device__ void runner_dopair4_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , float *pshift ) {
#else
__device__ void runner_dopair4_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , float *pshift ) {
#endif

    int k, pjd, ind, wrap_i, threadID;
    int pjoff;
    float4 pi[4], pj;
    int4 pot, pid, valid;
    float4 r2, ee, eff;
    float epot = 0.0f, dx[12], pjf[3], shift[3], w, *temp;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i < ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

    /* Get the wraps. */
    wrap_i = (count_i < cuda_frame) ? cuda_frame : count_i;
    
    /* Make sure everybody is in the same place. */
    __threadfence_block();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cjd );
        #else
            pj = parts_j[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        pj.x += shift[0]; pj.y += shift[1]; pj.z += shift[2];
        for ( k = 0 ; k < 3 ; k++ )
            pjf[k] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( ind = 0 ; ind < wrap_i ; ind += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pid.x = ind + threadID ) >= wrap_i )
                pid.x -= wrap_i;
            if ( ( pid.y = ind + threadID + 1 ) >= wrap_i )
                pid.y -= wrap_i;
            if ( ( pid.z = ind + threadID + 2 ) >= wrap_i )
                pid.z -= wrap_i;
            if ( ( pid.w = ind + threadID + 3 ) >= wrap_i )
                pid.w -= wrap_i;
                
            /* Get the particle pointers. */
            #ifdef PARTS_TEX
                pi[0] = ( valid.x = ( pid.x < count_i ) ) ? tex2D( tex_parts , pid.x , cid ) : pj;
                pi[1] = ( valid.y = ( pid.y < count_i ) && ( ind + 1 < wrap_i ) ) ? tex2D( tex_parts , pid.y , cid ) : pj;
                pi[2] = ( valid.z = ( pid.z < count_i ) && ( ind + 2 < wrap_i ) ) ? tex2D( tex_parts , pid.z , cid ) : pj;
                pi[3] = ( valid.w = ( pid.w < count_i ) && ( ind + 3 < wrap_i ) ) ? tex2D( tex_parts , pid.w , cid ) : pj;
            #else
                pi[0] = ( valid.x = ( pid.x < count_i ) ) ? parts_i[ pid.x] : pj;
                pi[1] = ( valid.y = ( pid.y < count_i ) && ( ind + 1 < wrap_i ) ) ? parts_i[ pid.y ] : pj;
                pi[2] = ( valid.z = ( pid.z < count_i ) && ( ind + 2 < wrap_i ) ) ? parts_i[ pid.z ] : pj;
                pi[3] = ( valid.w = ( pid.w < count_i ) && ( ind + 3 < wrap_i ) ) ? parts_i[ pid.w ] : pj;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pj.x - pi[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pj.y - pi[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pj.z - pi[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pj.x - pi[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pj.y - pi[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pj.z - pi[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pj.x - pi[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pj.y - pi[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pj.z - pi[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pj.x - pi[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pj.y - pi[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pj.z - pi[3].z; r2.w += dx[11] * dx[11];

                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? tex1D( tex_pind , pjoff + pi[0].w ) : 0;
            pot.y = valid.y ? tex1D( tex_pind , pjoff + pi[1].w ) : 0;
            pot.z = valid.z ? tex1D( tex_pind , pjoff + pi[2].w ) : 0;
            pot.w = valid.w ? tex1D( tex_pind , pjoff + pi[3].w ) : 0;
            
            /* if ( pot.x != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.y != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.z != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.w != 0 )
                atomicAdd( &cuda_rcount , 1 ); */
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Store the interaction energy. */
            epot += ee.x + ee.y + ee.z + ee.w;
            
            /* Update the forces. */
            if ( valid.x ) {
                pjf[0] -= ( w = eff.x * dx[0] ); forces_i[ 3*pid.x + 0 ] += w;
                pjf[1] -= ( w = eff.x * dx[1] ); forces_i[ 3*pid.x + 1 ] += w;
                pjf[2] -= ( w = eff.x * dx[2] ); forces_i[ 3*pid.x + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.y ) {
                pjf[0] -= ( w = eff.y * dx[3] ); forces_i[ 3*pid.y + 0 ] += w;
                pjf[1] -= ( w = eff.y * dx[4] ); forces_i[ 3*pid.y + 1 ] += w;
                pjf[2] -= ( w = eff.y * dx[5] ); forces_i[ 3*pid.y + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.z ) {
                pjf[0] -= ( w = eff.z * dx[6] ); forces_i[ 3*pid.z + 0 ] += w;
                pjf[1] -= ( w = eff.z * dx[7] ); forces_i[ 3*pid.z + 1 ] += w;
                pjf[2] -= ( w = eff.z * dx[8] ); forces_i[ 3*pid.z + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.w ) {
                pjf[0] -= ( w = eff.w * dx[9] ); forces_i[ 3*pid.w + 0 ] += w;
                pjf[1] -= ( w = eff.w * dx[10] ); forces_i[ 3*pid.w + 1 ] += w;
                pjf[2] -= ( w = eff.w * dx[11] ); forces_i[ 3*pid.w + 2 ] += w;
                }
            __threadfence_block();
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_j[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        __threadfence_block();
            
        } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_pair)
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair_verlet_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist ) {
#else
__device__ void runner_dopair_verlet_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist ) {
#endif

    int k, j, i, ind, jnd, pid, pjd, spid, spjd, pjdid, threadID, wrap, cj;
    int pioff;
    unsigned int swap_i, dmaxdist;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, r2, w, ee = 0.0f, eff = 0.0f, nshift, inshift;
    float dx[3], pif[3], shift[3], shiftn[3], *temp;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;
       
    TIMER_TIC2
        
    /* Re-build sorted pairs list? */
    if ( verlet_rebuild ) {
    
        /* Pack the parts of i and j into the sort arrays. */
        for ( k = threadID ; k < count_i ; k += cuda_frame ) {
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , k , cid );
            #else
                pi = parts_i[ k ];
            #endif
            sort_i[k] = ( k << 16 ) |
                (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
            }
        for ( k = threadID ; k < count_j ; k += cuda_frame ) {
            #ifdef PARTS_TEX
                pj = tex2D( tex_parts , k , cjd );
            #else
                pj = parts_j[ k ];
            #endif
            sort_j[k] = ( k << 16 ) | 
                (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj.x)*shiftn[0] + (shift[1]+pj.y)*shiftn[1] + (shift[2]+pj.z)*shiftn[2]) );
            }
            
        /* Make sure all the memory is in the right place. */
        __threadfence_block();
        
        /* Sort using normalized bitonic sort. */
        for ( k = 1 ; k < count_i ; k *= 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_i ; i += cuda_frame ) {
                jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
                if ( jnd < count_i && ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                    swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            for ( j = k/2 ; j > 0 ; j = j / 2 ) {
                for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_i ; i += cuda_frame ) {
                    jnd = ind + j;
                    if ( ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                        swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                        }
                    }
                __threadfence_block();
                }
            }
        for ( k = 1 ; k < count_j ; k *= 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_j ; i += cuda_frame ) {
                jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
                if ( jnd < count_j && ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                    swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            for ( j = k/2 ; j > 0 ; j = j / 2 ) {
                for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_j ; i += cuda_frame ) {
                    jnd = ind + j;
                    if ( ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                        swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                        }
                    }
                __threadfence_block();
                }
            }

        /* Store the sorted list back to global memory. */
        cuda_memcpy( sortlist , sort_i , sizeof(int) * count_i );
        cuda_memcpy( &sortlist[count_i] , sort_j , sizeof(int) * count_j );
            
        } /* re-build sorted pairs list. */
        
    /* Otherwise, just read it from memory. */
    else {
        cuda_memcpy( sort_i , sortlist , sizeof(int) * count_i );
        cuda_memcpy( sort_j , &sortlist[count_i] , sizeof(int) * count_j );
        __threadfence_block();
        }
        
    TIMER_TOC2(tid_sort)
        
        
    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dmaxdist )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd = pjdid + threadID ) >= wrap )
                pjd -= wrap;
            
            /* Do we have a pair? */
            if ( pjd < cj ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                spjd = sort_j[pjd] >> 16;
                #ifdef PARTS_TEX
                    pj = tex2D( tex_parts , spjd , cjd );
                #else
                    pj = parts_j[ spjd ];
                #endif

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
                dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
                dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];
                    
                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = tex1D( tex_pind , pioff + pj.w ) ) != 0 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    
                    /* Interact particles pi and pj. */
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pif[k] -= w;
                        forces_j[ 3*spjd + k ] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair4_verlet_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist ) {
#else
__device__ void runner_dopair4_verlet_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift , int verlet_rebuild , unsigned int *sortlist ) {
#endif

    int k, j, i, ind, jnd, pid, spid, pjdid, threadID, wrap, cj;
    int pioff;
    unsigned int swap_i, dmaxdist;
    float4 pi, pj[4];
    int4 pot, pjd, spjd, valid;
    float4 ee, eff, r2;
    float epot = 0.0f, w, nshift, inshift;
    float dx[12], pif[3], shift[3], shiftn[3], *temp;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }

        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dmaxdist = 2 + cuda_dscale * cuda_maxdist;
       
    TIMER_TIC2
        
    /* Re-build sorted pairs list? */
    if ( verlet_rebuild ) {
    
        /* Pack the parts of i and j into the sort arrays. */
        for ( k = threadID ; k < count_i ; k += cuda_frame ) {
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , k , cid );
            #else
                pi = parts_i[ k ];
            #endif
            sort_i[k] = ( k << 16 ) |
                (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
            }
        for ( k = threadID ; k < count_j ; k += cuda_frame ) {
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , k , cjd );
            #else
                pi = parts_j[ k ];
            #endif
            sort_j[k] = ( k << 16 ) | 
                (unsigned int)( cuda_dscale * (nshift + (shift[0]+pi.x)*shiftn[0] + (shift[1]+pi.y)*shiftn[1] + (shift[2]+pi.z)*shiftn[2]) );
            }
            
        /* Make sure all the memory is in the right place. */
        __threadfence_block();
        
        /* Sort using normalized bitonic sort. */
        for ( k = 1 ; k < count_i ; k *= 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_i ; i += cuda_frame ) {
                jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
                if ( jnd < count_i && ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                    swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            for ( j = k/2 ; j > 0 ; j = j / 2 ) {
                for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_i ; i += cuda_frame ) {
                    jnd = ind + j;
                    if ( ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                        swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                        }
                    }
                __threadfence_block();
                }
            }
        for ( k = 1 ; k < count_j ; k *= 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_j ; i += cuda_frame ) {
                jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
                if ( jnd < count_j && ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                    swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            for ( j = k/2 ; j > 0 ; j = j / 2 ) {
                for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_j ; i += cuda_frame ) {
                    jnd = ind + j;
                    if ( ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                        swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                        }
                    }
                __threadfence_block();
                }
            }

        /* Store the sorted list back to global memory. */
        cuda_memcpy( sortlist , sort_i , sizeof(int) * count_i );
        cuda_memcpy( &sortlist[count_i] , sort_j , sizeof(int) * count_j );
            
        } /* re-build sorted pairs list. */
        
    /* Otherwise, just read it from memory. */
    else {
        cuda_memcpy( sort_i , sortlist , sizeof(int) * count_i );
        cuda_memcpy( sort_j , &sortlist[count_i] , sizeof(int) * count_j );
        __threadfence_block();
        }
        
    TIMER_TOC2(tid_sort)
        
        
    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dmaxdist )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd.x = pjdid + threadID ) >= wrap )
                pjd.x -= wrap;
            if ( ( pjd.y = pjdid + threadID + 1 ) >= wrap )
                pjd.y -= wrap;
            if ( ( pjd.z = pjdid + threadID + 2 ) >= wrap )
                pjd.z -= wrap;
            if ( ( pjd.w = pjdid + threadID + 3 ) >= wrap )
                pjd.w -= wrap;
                
            /* Get the particle pointers. */
            spjd.x = sort_j[pjd.x] >> 16; spjd.y = sort_j[pjd.y] >> 16; spjd.z = sort_j[pjd.z] >> 16; spjd.w = sort_j[pjd.w] >> 16; 
            #ifdef PARTS_TEX
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? tex2D( tex_parts , spjd.x , cjd ) : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? tex2D( tex_parts , spjd.y , cjd ) : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? tex2D( tex_parts , spjd.z , cjd ) : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? tex2D( tex_parts , spjd.w , cjd ) : pi;
            #else
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? parts_j[ spjd.x ] : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? parts_j[ spjd.y ] : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? parts_j[ spjd.z ] : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? parts_j[ spjd.w ] : pi;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pi.x - pj[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pi.y - pj[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pi.z - pj[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pi.x - pj[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pi.y - pj[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pi.z - pj[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pi.x - pj[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pi.y - pj[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pi.z - pj[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pi.x - pj[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pi.y - pj[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pi.z - pj[3].z; r2.w += dx[11] * dx[11];
                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? tex1D( tex_pind , pioff + pj[0].w ) : 0;
            pot.y = valid.y ? tex1D( tex_pind , pioff + pj[1].w ) : 0;
            pot.z = valid.z ? tex1D( tex_pind , pioff + pj[2].w ) : 0;
            pot.w = valid.w ? tex1D( tex_pind , pioff + pj[3].w ) : 0;
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Store the interaction energy. */
            epot += ee.x + ee.y + ee.z + ee.w;
            
            /* Update the particle forces. */
            if ( valid.x ) {
                pif[0] -= ( w = eff.x * dx[0] ); forces_j[ 3*spjd.x + 0 ] += w;
                pif[1] -= ( w = eff.x * dx[1] ); forces_j[ 3*spjd.x + 1 ] += w;
                pif[2] -= ( w = eff.x * dx[2] ); forces_j[ 3*spjd.x + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.y ) {
                pif[0] -= ( w = eff.y * dx[3] ); forces_j[ 3*spjd.y + 0 ] += w;
                pif[1] -= ( w = eff.y * dx[4] ); forces_j[ 3*spjd.y + 1 ] += w;
                pif[2] -= ( w = eff.y * dx[5] ); forces_j[ 3*spjd.y + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.z ) {
                pif[0] -= ( w = eff.z * dx[6] ); forces_j[ 3*spjd.z + 0 ] += w;
                pif[1] -= ( w = eff.z * dx[7] ); forces_j[ 3*spjd.z + 1 ] += w;
                pif[2] -= ( w = eff.z * dx[8] ); forces_j[ 3*spjd.z + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.w ) {
                pif[0] -= ( w = eff.w * dx[9] ); forces_j[ 3*spjd.w + 0 ] += w;
                pif[1] -= ( w = eff.w * dx[10] ); forces_j[ 3*spjd.w + 1 ] += w;
                pif[2] -= ( w = eff.w * dx[11] ); forces_j[ 3*spjd.w + 2 ] += w;
                }
            __threadfence_block();
            
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair_sorted_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift ) {
#else
__device__ void runner_dopair_sorted_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift ) {
#endif

    int k, j, i, ind, jnd, pid, pjd, spid, spjd, pjdid, threadID, wrap, cj;
    int pioff, dcutoff;
    unsigned int swap_i;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, r2, w, ee = 0.0f, eff = 0.0f, nshift, inshift;
    float dx[3], pif[3], shift[3], shiftn[3], *temp;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dcutoff = 2 + cuda_dscale * cuda_cutoff;
       
    TIMER_TIC2
       
    /* Pack the parts of i and j into the sort arrays. */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , k , cid );
        #else
            pi = parts_i[ k ];
        #endif
        sort_i[k] = ( k << 16 ) |
            (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , k , cjd );
        #else
            pj = parts_j[ k ];
        #endif
        sort_j[k] = ( k << 16 ) | 
            (unsigned int)( cuda_dscale * (nshift + (shift[0]+pj.x)*shiftn[0] + (shift[1]+pj.y)*shiftn[1] + (shift[2]+pj.z)*shiftn[2]) );
        }
        
    /* Make sure all the memory is in the right place. */
    __threadfence_block();
    
    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count_i ; k *= 2 ) {
        for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_i ; i += cuda_frame ) {
            jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
            if ( jnd < count_i && ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                }
            }
        __threadfence_block();
        for ( j = k/2 ; j > 0 ; j = j / 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_i ; i += cuda_frame ) {
                jnd = ind + j;
                if ( ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                    swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            }
        }
    for ( k = 1 ; k < count_j ; k *= 2 ) {
        for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_j ; i += cuda_frame ) {
            jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
            if ( jnd < count_j && ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                }
            }
        __threadfence_block();
        for ( j = k/2 ; j > 0 ; j = j / 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_j ; i += cuda_frame ) {
                jnd = ind + j;
                if ( ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                    swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            }
        }
        
    TIMER_TOC2(tid_sort)
        

    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dcutoff )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd = pjdid + threadID ) >= wrap )
                pjd -= wrap;
            
            /* Do we have a pair? */
            if ( pjd < cj ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                spjd = sort_j[pjd] >> 16;
                #ifdef PARTS_TEX
                    pj = tex2D( tex_parts , spjd , cjd );
                #else
                    pj = parts_j[ spjd ];
                #endif

                /* Compute the radius between pi and pj. */
                r2 = 0.0f;
                dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
                dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
                dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];
                    
                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = tex1D( tex_pind , pioff + pj.w ) ) != 0 ) {

                    /* printf( "runner_dopair_cuda[%i]: doing pair [%i,%i] with r=%i (d=%i).\n" ,
                        threadID , sort_i[pid].ind , sort_j[pjd].ind , (int)(sqrtf(r2)*1000.0) , (int)((sort_j[pjd].d - sort_i[pid].d)*1000) ); */

                    // atomicAdd( &cuda_pairs_done , 1 );
                    
                    /* Interact particles pi and pj. */
                    potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pif[k] -= w;
                        forces_j[ 3*spjd + k ] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param icid Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param icjd Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 * @param cid Part buffer in local memory.
 * @param cjd Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_dopair4_sorted_cuda ( int cid , int count_i , int cjd , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift ) {
#else
__device__ void runner_dopair4_sorted_cuda ( float4 *parts_i , int count_i , float4 *parts_j , int count_j , float *forces_i , float *forces_j , unsigned int *sort_i , unsigned int *sort_j , float *pshift ) {
#endif

    int k, j, i, ind, jnd, pid, spid, pjdid, threadID, wrap, cj;
    int pioff, dcutoff;
    unsigned int swap_i;
    float4 pi, pj[4];
    int4 pot, pjd, spjd, valid;
    float4 ee, eff, r2;
    float epot = 0.0f, w, nshift, inshift;
    float dx[12], pif[3], shift[3], shiftn[3], *temp;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i > ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        #ifdef PARTS_TEX
            k = cid; cid = cjd; cjd = k;
        #else
            float4 *temp4 = parts_i; parts_i = parts_j; parts_j = temp4;
        #endif
        k = count_i; count_i = count_j; count_j = k;
        temp = forces_i; forces_i = forces_j; forces_j = temp;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
        
    /* Pre-compute the inverse norm of the shift. */
    nshift = sqrtf( shift[0]*shift[0] + shift[1]*shift[1] + shift[2]*shift[2] );
    inshift = 1.0f / nshift;
    shiftn[0] = inshift*shift[0]; shiftn[1] = inshift*shift[1]; shiftn[2] = inshift*shift[2];
    dcutoff = 2 + cuda_dscale * cuda_cutoff;
       
    TIMER_TIC2
       
    /* Pack the parts of i and j into the sort arrays. */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , k , cid );
        #else
            pi = parts_i[ k ];
        #endif
        sort_i[k] = ( k << 16 ) |
            (unsigned int)( cuda_dscale * (nshift + pi.x*shiftn[0] + pi.y*shiftn[1] + pi.z*shiftn[2]) );
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , k , cjd );
        #else
            pi = parts_j[ k ];
        #endif
        sort_j[k] = ( k << 16 ) | 
            (unsigned int)( cuda_dscale * (nshift + (shift[0]+pi.x)*shiftn[0] + (shift[1]+pi.y)*shiftn[1] + (shift[2]+pi.z)*shiftn[2]) );
        }
        
    /* Make sure all the memory is in the right place. */
    __threadfence_block();
    
    /* Sort using normalized bitonic sort. */
    for ( k = 1 ; k < count_i ; k *= 2 ) {
        for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_i ; i += cuda_frame ) {
            jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
            if ( jnd < count_i && ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                }
            }
        __threadfence_block();
        for ( j = k/2 ; j > 0 ; j = j / 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_i ; i += cuda_frame ) {
                jnd = ind + j;
                if ( ( sort_i[ind] & 0xffff ) < ( sort_i[jnd] & 0xffff ) ) {
                    swap_i = sort_i[ind]; sort_i[ind] = sort_i[jnd]; sort_i[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            }
        }
    for ( k = 1 ; k < count_j ; k *= 2 ) {
        for ( i = threadID ; ( ind = ( i & ~(k - 1) ) * 2 + ( i & (k - 1) ) ) < count_j ; i += cuda_frame ) {
            jnd = ( i & ~(k - 1) ) * 2 + 2*k - ( i & (k - 1) ) - 1;
            if ( jnd < count_j && ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                }
            }
        __threadfence_block();
        for ( j = k/2 ; j > 0 ; j = j / 2 ) {
            for ( i = threadID ; ( ind = ( i & ~(j - 1) ) * 2 + ( i & (j - 1) ) ) + j < count_j ; i += cuda_frame ) {
                jnd = ind + j;
                if ( ( sort_j[ind] & 0xffff ) > ( sort_j[jnd] & 0xffff ) ) {
                    swap_i = sort_j[ind]; sort_j[ind] = sort_j[jnd]; sort_j[jnd] = swap_i;
                    }
                }
            __threadfence_block();
            }
        }
        
    TIMER_TOC2(tid_sort)
        

    /* Loop over the particles in cell_j, frame-wise. */
    cj = count_j;
    for ( pid = threadID ; pid < count_i ; pid += cuda_frame ) {
    
        /* Get the wrap. */
        while ( cj > 0 && ( sort_j[cj-1] & 0xffff ) - ( sort_i[pid & ~(cuda_frame - 1)] & 0xffff ) > dcutoff )
            cj -= 1;
        if ( cj == 0 )
            break;
        else if ( cj < cuda_frame )
            wrap = max( cj , min( count_i - (pid & ~(cuda_frame - 1)) , cuda_frame ) );
        else
            wrap = cj;
            
        /* Get a direct pointer on the pjdth part in cell_j. */
        spid = sort_i[pid] >> 16;
        #ifdef PARTS_TEX
            pi = tex2D( tex_parts , spid , cid );
        #else
            pi = parts_i[ spid ];
        #endif
        pioff = pi.w * cuda_maxtype;
        pi.x -= shift[0]; pi.y -= shift[1]; pi.z -= shift[2];
        pif[0] = 0.0f; pif[1] = 0.0f; pif[2] = 0.0f;
        
        /* Loop over the particles in cell_i. */
        for ( pjdid = 0 ; pjdid < wrap ; pjdid += 4 ) {
        
            /* Wrap the particle index correctly. */
            if ( ( pjd.x = pjdid + threadID ) >= wrap )
                pjd.x -= wrap;
            if ( ( pjd.y = pjdid + threadID + 1 ) >= wrap )
                pjd.y -= wrap;
            if ( ( pjd.z = pjdid + threadID + 2 ) >= wrap )
                pjd.z -= wrap;
            if ( ( pjd.w = pjdid + threadID + 3 ) >= wrap )
                pjd.w -= wrap;
                
            /* Get the particle pointers. */
            spjd.x = sort_j[pjd.x] >> 16; spjd.y = sort_j[pjd.y] >> 16; spjd.z = sort_j[pjd.z] >> 16; spjd.w = sort_j[pjd.w] >> 16; 
            #ifdef PARTS_TEX
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? tex2D( tex_parts , spjd.x , cjd ) : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? tex2D( tex_parts , spjd.y , cjd ) : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? tex2D( tex_parts , spjd.z , cjd ) : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? tex2D( tex_parts , spjd.w , cjd ) : pi;
            #else
                pj[0] = ( valid.x = ( pjd.x < cj ) ) ? parts_j[ spjd.x ] : pi;
                pj[1] = ( valid.y = ( pjd.y < cj ) && ( pjdid + 1 < wrap ) ) ? parts_j[ spjd.y ] : pi;
                pj[2] = ( valid.z = ( pjd.z < cj ) && ( pjdid + 2 < wrap ) ) ? parts_j[ spjd.z ] : pi;
                pj[3] = ( valid.w = ( pjd.w < cj ) && ( pjdid + 3 < wrap ) ) ? parts_j[ spjd.w ] : pi;
            #endif
            
            /* Compute the pairwise distances. */
            r2 = make_float4( 0.0f );
            dx[0] = pi.x - pj[0].x; r2.x += dx[0] * dx[0];
            dx[1] = pi.y - pj[0].y; r2.x += dx[1] * dx[1];
            dx[2] = pi.z - pj[0].z; r2.x += dx[2] * dx[2];
            dx[3] = pi.x - pj[1].x; r2.y += dx[3] * dx[3];
            dx[4] = pi.y - pj[1].y; r2.y += dx[4] * dx[4];
            dx[5] = pi.z - pj[1].z; r2.y += dx[5] * dx[5];
            dx[6] = pi.x - pj[2].x; r2.z += dx[6] * dx[6];
            dx[7] = pi.y - pj[2].y; r2.z += dx[7] * dx[7];
            dx[8] = pi.z - pj[2].z; r2.z += dx[8] * dx[8];
            dx[9] = pi.x - pj[3].x; r2.w += dx[9] * dx[9];
            dx[10] = pi.y - pj[3].y; r2.w += dx[10] * dx[10];
            dx[11] = pi.z - pj[3].z; r2.w += dx[11] * dx[11];
                
            /* Get the potentials. */
            valid.x = ( valid.x && r2.x < cuda_cutoff2 );
            valid.y = ( valid.y && r2.y < cuda_cutoff2 );
            valid.z = ( valid.z && r2.z < cuda_cutoff2 );
            valid.w = ( valid.w && r2.w < cuda_cutoff2 );
            pot.x = valid.x ? tex1D( tex_pind , pioff + pj[0].w ) : 0;
            pot.y = valid.y ? tex1D( tex_pind , pioff + pj[1].w ) : 0;
            pot.z = valid.z ? tex1D( tex_pind , pioff + pj[2].w ) : 0;
            pot.w = valid.w ? tex1D( tex_pind , pioff + pj[3].w ) : 0;
            
            /* if ( pot.x != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.y != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.z != 0 )
                atomicAdd( &cuda_rcount , 1 );
            if ( pot.w != 0 )
                atomicAdd( &cuda_rcount , 1 ); */
            
            /* Compute the interaction. */
            potential_eval4_cuda_tex( pot , r2 , &ee , &eff );
            
            /* Store the interaction energy. */
            epot += ee.x + ee.y + ee.z + ee.w;
            
            /* Update the particle forces. */
            if ( valid.x ) {
                pif[0] -= ( w = eff.x * dx[0] ); forces_j[ 3*spjd.x + 0 ] += w;
                pif[1] -= ( w = eff.x * dx[1] ); forces_j[ 3*spjd.x + 1 ] += w;
                pif[2] -= ( w = eff.x * dx[2] ); forces_j[ 3*spjd.x + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.y ) {
                pif[0] -= ( w = eff.y * dx[3] ); forces_j[ 3*spjd.y + 0 ] += w;
                pif[1] -= ( w = eff.y * dx[4] ); forces_j[ 3*spjd.y + 1 ] += w;
                pif[2] -= ( w = eff.y * dx[5] ); forces_j[ 3*spjd.y + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.z ) {
                pif[0] -= ( w = eff.z * dx[6] ); forces_j[ 3*spjd.z + 0 ] += w;
                pif[1] -= ( w = eff.z * dx[7] ); forces_j[ 3*spjd.z + 1 ] += w;
                pif[2] -= ( w = eff.z * dx[8] ); forces_j[ 3*spjd.z + 2 ] += w;
                }
            __threadfence_block();
            if ( valid.w ) {
                pif[0] -= ( w = eff.w * dx[9] ); forces_j[ 3*spjd.w + 0 ] += w;
                pif[1] -= ( w = eff.w * dx[10] ); forces_j[ 3*spjd.w + 1 ] += w;
                pif[2] -= ( w = eff.w * dx[11] ); forces_j[ 3*spjd.w + 2 ] += w;
                }
            __threadfence_block();
            
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces_i[ 3*spid + k ] += pif[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    TIMER_TOC(tid_pair)
    
    }


/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_doself_cuda ( int cid , int count , float *forces ) {
#else
__device__ void runner_doself_cuda ( float4 *parts , int count , float *forces ) {
#endif

    int k, pid, pjd, threadID;
    int pjoff;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, dx[3], pjf[3], r2, w, ee, eff;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Make sure everybody is in the same place. */
    __threadfence_block();

    /* Loop over the particles in the cell, frame-wise. */
    for ( pjd = threadID ; pjd < count-1 ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        #ifdef PARTS_TEX
            pj = tex2D( tex_parts , pjd , cid );
        #else
            pj = parts_j[ pjd ];
        #endif
        pjoff = pj.w * cuda_maxtype;
        pjf[0] = 0.0f; pjf[1] = 0.0f; pjf[2] = 0.0f;
            
        /* Loop over the particles in cell_i. */
        for ( pid = pjd+1 ; pid < count ; pid++ ) {
        
            /* Get a handle on the wrapped particle pid in cell_i. */
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , pid , cid );
            #else
                pi = parts_i[ pid ];
            #endif

            /* Compute the radius between pi and pj. */
            r2 = 0.0f;
            dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
            dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
            dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];

            /* Set the null potential if anything is bad. */
            if ( r2 < cuda_cutoff2 && ( pot = tex1D( tex_pind , pjoff + pi.w ) ) != 0 ) {

                // atomicAdd( &cuda_pairs_done , 1 );
            
                /* Interact particles pi and pj. */
                potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                /* Store the interaction force and energy. */
                epot += ee;
                for ( k = 0 ; k < 3 ; k++ ) {
                    w = eff * dx[k];
                    forces[ 3*pid + k ] -= w;
                    pjf[k] += w;
                    }

                /* Sync the shared memory values. */
                __threadfence_block();
            
                } /* in range and potential. */

            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            forces[ 3*pjd + k ] += pjf[k];

        /* Sync the shared memory values. */
        __threadfence_block();

        } /* loop over the particles in cell_j. */
        
    TIMER_TOC(tid_self)
    
    }
    
    
/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 * @param parts Part buffer in local memory.
 *
 * @sa #runner_dopair.
 */
 
#ifdef PARTS_TEX
__device__ void runner_doself_diag_cuda ( int cid , int count , float *forces ) {
#else
__device__ void runner_doself_diag_cuda ( float4 *parts , int count , float *forces ) {
#endif

    int diag, k, diag_max, step, pid, pjd, threadID;
    unsigned int packed;
    float4 pi, pj;
    int pot;
    float epot = 0.0f, dx[3], r2, w[3], ee, eff;
    
    TIMER_TIC
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Step along the number of diagonal entries. */
    diag_max = count * (count - 1) / 2; step = 1;
    for ( diag = 0 ; diag < diag_max ; diag += step ) {
    
        /* is it time for this thread to step in? */
        if ( diag == threadID ) {
            step = diag;
            diag = (diag + 2) * (diag + 1) / 2 - 1;
            }
            
        /* If running, continue with the interactions. */
        if ( diag >= threadID && diag < diag_max ) {
        
            /* Increase the step if necessary. */
            if ( step < cuda_frame )
                step += 1;
    
            /* Get the location of the kth entry on the diagonal. */
            packed = tex1D( tex_diags , diag );
            pid = packed >> 16; pjd = count - (packed & 0xffff);
            
            /* Get a handle on the particles. */
            #ifdef PARTS_TEX
                pi = tex2D( tex_parts , pid , cid );
                pj = tex2D( tex_parts , pjd , cid );
            #else
                pi = parts[ pid ];
                pj = parts[ pjd ];
            #endif

            /* Compute the radius between pi and pj. */
            r2 = 0.0f;
            dx[0] = pi.x - pj.x; r2 += dx[0]*dx[0];
            dx[1] = pi.y - pj.y; r2 += dx[1]*dx[1];
            dx[2] = pi.z - pj.z; r2 += dx[2]*dx[2];

            /* Set the null potential if anything is bad. */
            if ( r2 < cuda_cutoff2 && ( pot = tex1D( tex_pind , pj.w*cuda_maxtype + pi.w ) ) != 0 ) {

                // atomicAdd( &cuda_rcount , 1 );
                    
                /* Interact particles pi and pj. */
                potential_eval_cuda_tex( pot , r2 , &ee , &eff );

                /* Store the interaction force on pi and energy. */
                for ( k = 0 ; k < 3 ; k++ ) {
                    w[k] = eff * dx[k];
                    forces[ 3*pid + k ] -= w[k];
                    }

                /* Sync the shared memory values. */
                __threadfence_block();

                /* Store the interaction force on pj. */
                epot += ee;
                for ( k = 0 ; k < 3 ; k++ )
                    forces[ 3*pjd + k ] += w[k];

                /* Sync the shared memory values. */
                __threadfence_block();

                } /* range and potential? */

            /* printf( "runner_doself_diag_cuda[%i]: diag=%i, step=%i, i=%i, j=%i.\n" ,
                threadID , diag , step , pid , pjd ); */

            } /* is it this thread's turn? */
    
        } /* Loop over diagonal indices. */
        
    TIMER_TOC(tid_self)
    
    }
    
    
/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus not
 * be externalized.
 */
 
int runner_bind ( hipArray *cuArray_coeffs , hipArray *cuArray_alphas , hipArray *cuArray_pind , hipArray *cuArray_diags ) {

    /* Bind the coeffs. */
    cuda_coeffs = cuArray_coeffs;
    if ( hipBindTextureToArray( tex_coeffs , cuArray_coeffs ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Bind the alphas. */
    cuda_alphas = cuArray_alphas;
    if ( hipBindTextureToArray( tex_alphas , cuArray_alphas ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Bind the pinds. */
    cuda_pind = cuArray_pind;
    if ( hipBindTextureToArray( tex_pind , cuArray_pind ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Bind the diags. */
    cuda_diags = cuArray_diags;
    if ( hipBindTextureToArray( tex_diags , cuArray_diags ) != hipSuccess )
        return cuda_error(engine_err_cuda);
        
    /* Rock and roll. */
    return runner_err_ok;

    }


/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus not
 * be externalized.
 */
 
int runner_parts_bind ( hipArray *cuArray_parts ) {

    /* Bind the coeffs. */
    if ( hipBindTextureToArray( tex_parts , cuArray_parts ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Rock and roll. */
    return runner_err_ok;

    }


/**
 * @brief Bind textures to the given cuda Arrays.
 *
 *
 * Hack to get around the fact that textures are static and can thus not
 * be externalized.
 */
 
int runner_parts_unbind ( ) {

    /* Bind the coeffs. */
    if ( hipUnbindTexture( tex_parts ) != hipSuccess )
        return cuda_error(engine_err_cuda);
    
    /* Rock and roll. */
    return runner_err_ok;

    }


/** This set of defines and includes produces kernels with buffers for multiples
 *  of 32 particles up to 512 cuda_maxparts.
 */
 
#define cuda_nrparts 32
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 64
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 96
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 128
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 160
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 192
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 224
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 256
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 288
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 320
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 352
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 384
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 416
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 448
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 480
    #include "runner_cuda_main.h"
#undef cuda_nrparts

#define cuda_nrparts 512
    #include "runner_cuda_main.h"





