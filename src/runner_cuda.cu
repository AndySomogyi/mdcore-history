#include "hip/hip_runtime.h"
/*******************************************************************************
 * This file is part of mdcore.
 * Coypright (c) 2012 Pedro Gonnet (gonnet@maths.ox.ac.uk)
 * 
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published
 * by the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU Lesser General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * 
 ******************************************************************************/

/* Include configuration header */
#include "../config.h"

/* Include some standard header files */
#include <stdlib.h>
#include <stdio.h>
#include <pthread.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <limits.h>

/* Include some conditional headers. */
#ifdef HAVE_MPI
    #include <mpi.h>
#endif

/* Force single precision. */
#define FPTYPE_SINGLE 1

/* Include local headers */
#include "cycle.h"
#include "errs.h"
#include "fptype.h"
#include "part.h"
#include "cell.h"
#include "space.h"
#include "potential.h"
#include "engine.h"
#include "runner.h"


/* Set the max number of parts for shared buffers. */
#define cuda_maxparts 160
#define cuda_frame 32


/* The constant null potential. */
__constant__ struct potential *potential_null_cuda = NULL;

/* The number of cells and pairs. */
__constant__ int cuda_nr_pairs = 0;
__constant__ int cuda_nr_cells = 0;

/* The mutex for accessing the cell pair list. */
__device__ int cuda_cell_mutex = 0;

/* The list of cell pairs. */
__device__ struct cellpair_cuda *cuda_pairs;
__device__ int *cuda_taboo;

/* The index of the next free cell pair. */
__device__ int cuda_pair_next = 0;

/* Some constants. */
__constant__ float cuda_cutoff2 = 0.0f;
__constant__ struct potential **cuda_p;
__constant__ int cuda_maxtype = 0;

/* The potential coefficients, as a texture. */
texture< float , hipTextureType1D > tex_pots;


/**
 * @brief Lock a device mutex.
 *
 * @param m The mutex.
 *
 * Loops until the mutex can be set. Note that only one thread
 * can do this at a time, so to synchronize blocks, only a single thread of
 * each block should call it.
 */

__device__ inline void cuda_mutex_lock ( int *m ) {
    while ( atomicCAS( m , 0 , 1 ) != 0 );
    }


/**
 * @brief Unlock a device mutex.
 *
 * @param m The mutex.
 *
 * Does not check if the mutex had been locked.
 */

__device__ inline void cuda_mutex_unlock ( int *m ) {
    atomicExch( m , 0 );
    }


/** 
 * @brief Evaluates the given potential at the given point (interpolated).
 *
 * @param p The #potential to be evaluated.
 * @param r2 The radius at which it is to be evaluated, squared.
 * @param e Pointer to a floating-point value in which to store the
 *      interaction energy.
 * @param f Pointer to a floating-point value in which to store the
 *      magnitude of the interaction force divided by r.
 *
 * Note that for efficiency reasons, this function does not check if any
 * of the parameters are @c NULL or if @c sqrt(r2) is within the interval
 * of the #potential @c p.
 */

__device__ inline void potential_eval_cuda ( struct potential *p , float r2 , float *e , float *f ) {

    int ind, k;
    float x, ee, eff, *c, r;
    
    /* Get r for the right type. */
    r = sqrtf(r2);
    
    /* compute the interval index */
    ind = fmaxf( 0.0f , p->alpha[0] + r * (p->alpha[1] + r * p->alpha[2]) );
    
    /* get the table offset */
    c = &(p->c[ind * potential_chunk]);
    
    /* adjust x to the interval */
    x = (r - c[0]) * c[1];
    
    /* compute the potential and its derivative */
    ee = c[2] * x + c[3];
    eff = c[2];
    for ( k = 4 ; k < potential_chunk ; k++ ) {
        eff = eff * x + ee;
        ee = ee * x + c[k];
        }

    /* store the result */
    *e = ee; *f = eff * c[1] / r;
        
    }


/**
 * @brief Compute the pairwise interactions for the given pair on a CUDA device.
 *
 * @param iparts_i Array of parts in the first cell.
 * @param count_i Number of parts in the first cell.
 * @param iparts_j Array of parts in the second cell.
 * @param count_j Number of parts in the second cell.
 * @param pshift A pointer to an array of three floating point values containing
 *      the vector separating the centers of @c cell_i and @c cell_j.
 *
 * @sa #runner_dopair.
 */
 
__device__ void runner_dopair_cuda ( struct part *iparts_i , int count_i , struct part *iparts_j , int count_j, float *pshift ) {

    int k, pid, pjd, ind, wrap, threadID;
    int pjoff;
    struct part_cuda *pi, *pj;
    struct part *temp;
    struct potential *pot;
    float epot = 0.0f, dx[3], pjx[3], pjf[3], shift[3], r2, w, ee, eff;
    __shared__ struct part_cuda parts_i[ cuda_maxparts ], parts_j[ cuda_maxparts ];
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Swap cells? cell_j loops in steps of frame... */
    if ( ( ( count_i + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_i < ( ( count_j + (cuda_frame-1) ) & ~(cuda_frame-1) ) - count_j ) {
        temp = iparts_i; iparts_i = iparts_j; iparts_j = temp;
        k = count_i; count_i = count_j; count_j = k;
        shift[0] = -pshift[0]; shift[1] = -pshift[1]; shift[2] = -pshift[2];
        }
    else {
        shift[0] = pshift[0]; shift[1] = pshift[1]; shift[2] = pshift[2];
        }
    
    /* Copy the particle data to the local buffers */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        parts_i[k].x[0] = iparts_i[k].x[0];
        parts_i[k].x[1] = iparts_i[k].x[1];
        parts_i[k].x[2] = iparts_i[k].x[2];
        parts_i[k].f[0] = iparts_i[k].f[0];
        parts_i[k].f[1] = iparts_i[k].f[1];
        parts_i[k].f[2] = iparts_i[k].f[2];
        parts_i[k].type = iparts_i[k].type;
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        parts_j[k].x[0] = iparts_j[k].x[0];
        parts_j[k].x[1] = iparts_j[k].x[1];
        parts_j[k].x[2] = iparts_j[k].x[2];
        parts_j[k].f[0] = iparts_j[k].f[0];
        parts_j[k].f[1] = iparts_j[k].f[1];
        parts_j[k].f[2] = iparts_j[k].f[2];
        parts_j[k].type = iparts_j[k].type;
        }
        
    /* Get the wrap. */
    if ( ( wrap = count_i ) < cuda_frame )
        wrap = cuda_frame;
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Loop over the particles in cell_j, frame-wise. */
    for ( pjd = threadID ; pjd < count_j ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = &parts_j[pjd];
        pjoff = pj->type * cuda_maxtype;
        for ( k = 0 ; k < 3 ; k++ ) {
            pjx[k] = pj->x[k] + shift[k];
            pjf[k] = 0.0f;
            }
        
        /* Loop over the particles in cell_i. */
        for ( pid = 0 ; pid < count_i ; pid++ ) {
        
            /* Wrap the particle index correctly. */
            if ( ( ind = pid + threadID ) >= wrap )
                ind -= wrap;
            
            /* Do we have a pair? */
            if ( ind < count_i ) {
            
                /* Get a handle on the wrapped particle pid in cell_i. */
                pi = &parts_i[ ind ];
                // printf( "runner_dopair_cuda: doing pair [%i,%i].\n" , pjd , ind );

                /* Compute the radius between pi and pj. */
                for ( r2 = 0.0f , k = 0 ; k < 3 ; k++ ) {
                    dx[k] = pi->x[k] - pjx[k];
                    r2 += dx[k] * dx[k];
                    }

                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = cuda_p[ pjoff + pi->type ] ) != NULL ) {

                    /* Interact particles pi and pj. */
                    potential_eval_cuda( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pi->f[k] -= w;
                        pjf[k] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();
                
                    } /* in range and potential. */

                } /* do we have a pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            pj->f[k] += pjf[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();
        
        } /* loop over the particles in cell_j. */
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Copy the particle data back from the local buffers */
    for ( k = threadID ; k < count_i ; k += cuda_frame ) {
        iparts_i[k].f[0] = parts_i[k].f[0];
        iparts_i[k].f[1] = parts_i[k].f[1];
        iparts_i[k].f[2] = parts_i[k].f[2];
        }
    for ( k = threadID ; k < count_j ; k += cuda_frame ) {
        iparts_j[k].f[0] = parts_j[k].f[0];
        iparts_j[k].f[1] = parts_j[k].f[1];
        iparts_j[k].f[2] = parts_j[k].f[2];
        }
        
    }


/**
 * @brief Compute the self interactions for the given cell on a CUDA device.
 *
 * @param iparts Array of parts in this cell.
 * @param count Number of parts in the cell.
 *
 * @sa #runner_dopair.
 */
 
__device__ void runner_doself_cuda ( struct part *iparts , int count ) {

    int k, ind, wrap, pid, pjd, threadID;
    int pjoff;
    struct part_cuda *pi, *pj;
    struct potential *pot;
    float epot = 0.0f, dx[3], pjx[3], pjf[3], r2, w, ee, eff;
    __shared__ struct part_cuda parts[ cuda_maxparts ];
    
    /* Get the size of the frame, i.e. the number of threads in this block. */
    threadID = threadIdx.x % cuda_frame;
    
    /* Copy the particle data to the local buffers */
    for ( k = threadID ; k < count ; k += cuda_frame ) {
        parts[k].x[0] = iparts[k].x[0];
        parts[k].x[1] = iparts[k].x[1];
        parts[k].x[2] = iparts[k].x[2];
        parts[k].f[0] = iparts[k].f[0];
        parts[k].f[1] = iparts[k].f[1];
        parts[k].f[2] = iparts[k].f[2];
        parts[k].type = iparts[k].type;
        }
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Loop over the particles in the cell, frame-wise. */
    for ( pjd = threadID ; pjd < count ; pjd += cuda_frame ) {
    
        /* Get a direct pointer on the pjdth part in cell_j. */
        pj = &parts[pjd];
        pjoff = pj->type * cuda_maxtype;
        for ( k = 0 ; k < 3 ; k++ ) {
            pjx[k] = pj->x[k];
            pjf[k] = 0.0f;
            }
            
        /* Set the wrapping. */
        wrap = (pjd + (cuda_frame - 1)) & ~(cuda_frame - 1);
        
        /* Loop over the particles in cell_i. */
        for ( pid = 0 ; pid < wrap ; pid++ ) {
        
            /* Get the correct wrapped id. */
            if ( ( ind = pid + threadID ) >= wrap )
                ind -= wrap;
                
            /* Valid particle pair? */
            if ( ind < pjd ) {
                
                // if ( threadID == 0 )
                // printf( "runner_doself_cuda: doing pair [%i,%i].\n" , pjd , ind );

                /* Get a handle on the wrapped particle pid in cell_i. */
                pi = &parts[ ind ];

                /* Compute the radius between pi and pj. */
                for ( r2 = 0.0f , k = 0 ; k < 3 ; k++ ) {
                    dx[k] = pi->x[k] - pjx[k];
                    r2 += dx[k] * dx[k];
                    }

                /* Set the null potential if anything is bad. */
                if ( r2 < cuda_cutoff2 && ( pot = cuda_p[ pjoff + pi->type ] ) != NULL ) {

                    /* Interact particles pi and pj. */
                    potential_eval_cuda( pot , r2 , &ee , &eff );

                    /* Store the interaction force and energy. */
                    epot += ee;
                    for ( k = 0 ; k < 3 ; k++ ) {
                        w = eff * dx[k];
                        pi->f[k] -= w;
                        pjf[k] += w;
                        }

                    /* Sync the shared memory values. */
                    __threadfence_block();

                    } /* range and potential? */

                } /* valid particle pair? */
        
            } /* loop over parts in cell_i. */
            
        /* Update the force on pj. */
        for ( k = 0 ; k < 3 ; k++ )
            pj->f[k] += pjf[k];
    
        /* Sync the shared memory values. */
        __threadfence_block();

        } /* loop over the particles in cell_j. */
    
    /* Make sure everybody is in the same place. */
    __syncthreads();

    /* Copy the particle data back from the local buffers */
    for ( k = threadID ; k < count ; k += cuda_frame ) {
        iparts[k].f[0] = parts[k].f[0];
        iparts[k].f[1] = parts[k].f[1];
        iparts[k].f[2] = parts[k].f[2];
        }
        
    }


/**
 * @brief Loop over the cell pairs and process them.
 *
 * @param cells Array of cells on the device.
 *
 */
 
__global__ void runner_run_cuda ( struct part *parts[] , int *counts ) {

    int blockID, threadID;
    int i;
    struct cellpair_cuda temp;
    __shared__ int finger, cid, cjd;
    
    /* Get the block and thread ids. */
    blockID = threadIdx.y;
    threadID = threadIdx.x;
    
    /* Check that we've got the correct warp size! */
    if ( warpSize != cuda_frame ) {
        if ( blockID == 0 && threadID == 0 )
            printf( "runner_run_cuda: error: the warp size of the device (%i) does not match the warp size mdcore was compiled for (%i).\n" ,
                warpSize , cuda_frame );
        return;
        }
    
    /* Greetings, earthling. */
    // if ( threadID == 0 )
        printf( "runner_run_cuda: thread %i of block %i says hi.\n" , threadID , blockID );
    
    /* If I'm the first thread in the first block, re-set the next pair. */
    if ( blockID == 0 && threadID == 0 )
        cuda_pair_next = 0;
        
    /* Make sure everybody is on the same page. */
    __threadfence();
            
    /* Main loop... */
    while ( cuda_pair_next < cuda_nr_pairs ) {
    
        /* Try to catch a pair. */
        if ( threadID == 0 ) {
        
            /* Lock the mutex. */
            cuda_mutex_lock( &cuda_cell_mutex );
            
            /* Loop over the remaining pairs... */
            for ( i = cuda_pair_next ; i < cuda_nr_pairs ; i++ )
                if ( cuda_taboo[ cuda_pairs[i].i ] == 0 &&
                     cuda_taboo[ cuda_pairs[i].j ] == 0 )
                    break;
                    
            /* If we actually got a pair, flip it to the top and decrease
               cuda_pair_next. */
            if ( i < cuda_nr_pairs ) {
                temp = cuda_pairs[i];
                cuda_pairs[i] = cuda_pairs[ cuda_pair_next ];
                cuda_pairs[ cuda_pair_next ] = temp;
                finger = cuda_pair_next;
                cid = cuda_pairs[finger].i; cjd = cuda_pairs[finger].j;
                cuda_pair_next += 1;
                cuda_taboo[ cid ] = 1;
                cuda_taboo[ cjd ] = 1;
                }
            else
                finger = -1;
            
            /* Make sure everybody is on the same page. */
            __threadfence();
        
            /* Un-lock the mutex. */
            cuda_mutex_unlock( &cuda_cell_mutex );
            
            }
            
        /* Get everybody together. */
        __syncthreads();
            
        /* If we actually got a pair, do it! */
        if ( finger >= 0 ) {
        
            if ( threadID == 0 )
                printf( "runner_run_cuda: block %i working on pair [%i,%i] (finger = %i).\n" , blockID , cid , cjd , finger );
        
            /* Do the pair. */
            if ( cid != cjd )
                runner_dopair_cuda( parts[cid] , counts[cid] , parts[cjd] , counts[cjd] , cuda_pairs[finger].shift );
            else
                runner_doself_cuda( parts[cid] , counts[cid] );
        
            /* Release the cells in the taboo list. */
            if ( threadID == 0 ) {
                cuda_taboo[ cid ] = 0;
                cuda_taboo[ cjd ] = 0;
                }
            
            /* Make sure everybody is on the same page. */
            __threadfence();
        
            }
    
        /* Get everybody together. */
        __syncthreads();
            
        } /* main loop. */

    }

